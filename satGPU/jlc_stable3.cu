#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#if !(defined(_WIN32) || defined(__WIN32__) || defined(WIN32))
#include <sys/time.h>
#include <sys/types.h>
#else
 #ifndef uint
 typedef unsigned int uint;
 #endif
#endif
#include <vector>
using namespace std;


#if (defined(_WIN32) || defined(__WIN32__) || defined(WIN32))
clock_t global_time;
#else
struct timeval time_start,time_stop;
#endif

//INITandy
// strategie per la variable selection in CPU (mask_prop())
#define NONE -1
#define MINCLAUSE 0
#define JEROWANGTWOSIDED 1
#define JEROWANGONESIDED 2
#define DLIS 3
#define DLCS 4
#define DEFAULT_VARSTRAT MINCLAUSE
//opzione per il sorting delle variabili unknown prima di lanciare il kernel
// nel mode 2. Si utilizza la euristica usata per la variable selection in CPU.
#define MAPSVARSORT_NO 0
#define MAPSVARSORT_YES 1
#define DEFAULT_MAPSVARSORT MAPSVARSORT_NO
//FINEandy


//*** IMPORTANT KERNEL STATIC OPTION.
//*** UnComment/Comment the following line
// In the first case (parameter defined) it simply assigns the block variables.
// In the second case it performs fast unit propagation after any assignment
#define SIMPLEBLOCK 0

//**** STATIC SIZE FOR block_vars and delta_vars
//**** ARRAYS USED IN PARALLELIZING SEARCH  // erano 101 e 110
#define  BV_SIZE 171 
#define  DV_SIZE 170 
#define  DV_SIZE_W 500 

//*** Comment/Uncomment the following line for information
//*** on memory allocation during CUDA execution
//#define MYTRACE 0

// The following PARAMETERS decide the parallelism degree
// and the complexity of the formula to be delagetd to CUDA
// We allow two versions, with static (#define) and dynamic values

//INITandy
int NUMPARAMETERS = 8; 
//#define TRACESELECT 0
//FINEandy

// *** PARAMETER #1 (argv 1)
// *** Modes for using GPU. Currently 0-3
int USEGPU = 0;   // The default value 

// *** PARAMETER #2 (argv 2)
// NUMBER OF BLOCKS = CUDABLOCKS
// LOG_BLKS is the number of vars fixed by the block address
int LOG_BLKS=2;
int CUDABLOCKS=(1<<LOG_BLKS); 

// *** PARAMETER #3 (argv 3)
// NUMBER OF THREADS per BLOCK  = CUDATHREADS 
// LOG_THRDS is the number of vars fixed by the thread
// In the current GPUs 10 is the maximum value (1024 threads per block)
int LOG_THRDS=5; 
int CUDATHREADS=(1<<LOG_THRDS); 

// *** PARAMETER #4 (argv 4)
// MaxV is the max number of non ground variables delegated to the GPU
int MaxV=50;  

//INITandy
// *** PARAMETER #6 (argv 6)
//strategia per select delle variabili
int strategia_varSelHost = DEFAULT_VARSTRAT;
// gestione delle strategie:
uint * host_strat_vars_weight; // "pesi" delle variabili (unknown)
int * host_strat_vars_Oneocc; // un idx di una delle clausole contententi la variabile
uint * host_strat_vars_clausesize; // dimensione della clausola (num. variabili unknown)

// *** PARAMETER #7 (argv 7)
// flag per attivare il riordono delle var prima di invocare il kernel in mode 2
int strategia_maps_varSort = DEFAULT_MAPSVARSORT;
// vettore di permutazione delle variabili (usato in filter_formula()):
int * permutator_maps_vars;
//FINEandy



// Delta is the number of variables treated ND by a CUDA thread. 
// The "+1" is needed since delta_vars[0] is not used!
int Delta= MaxV - LOG_BLKS + 1;   

// *** STATIC LIMITS TO FORMULA SIZE (before and after learning)

int MAX_NL   = 8000000;  
int MAX_NC   = 550000;
int nclausel = 0;

// *** PARAMETERS FOR FOR PARALLELIZING UNIT PROPAGATION
//   THREADS  (typically, 256, 512, 1024)
#define THREADS 512
//  Number of blocks used (assigned later using instance and HW data)
int PARblocks=1;          



//*********************************************
// GLOBAL ARRAYS USED
//*********************************************

int * formula;            // Formula and vars in the host    
int * clause_pointer;     // 
int * filtered_formula;   // Reduced formula in the tail 
int * filtered_clause_pointer; // stage
int * dev_formula;        // Formula to be passed in the device
int * dev_cp;             // in the UP 
int * dev_mapped_formula; // Formula in the device
int * dev_mapped_cp;      // in the unit propagation
int * host_vars;          // variables in the host
int * filtered_vars;      // Device vars (tail)
int * dev_vars;           // Device vars (u.p.)
int * dev_parma_vars;     // Device vars (u.p.)

// mode 5
int * dev_vars_w;         // unit watched
int * dev_v2c_w;         // unit watched
int * dev_w_w;
int * dev_next_w;
int * dev_unit_w;
int * dev_c2v_w;
int * dev_varstack_w;

//*********************************************

int * mask_data; //*** unificato per unico trasferimento
int * h_mask_data; 
int * maps_vars;

// *** stack/trail information

int * level;
int * learnt_clause;
int * seen;
int * refs;
int * trail;

int learning      =1;
int level_to_jump = -2; // in caso di backtracking/jumping

// *** GLOBAL VARIABLES

int NV,NC,NL;        // Formula parameters
int CUDA_count=0;    // It counts the calls to a device
int CUDA_count_lower=0;    // It counts the calls to a device (option 1 and 3)
int backtracking=0;  // It counts the backtrackings in the host
int backjump=0;  // It counts the backtrackings in the host
int sat_val,clauind,selected_var; // Again, for backtracking purposes
float deltatime;
hipEvent_t start, stop;

//*** Error Handling
__global__ void parmask_prop( int* mask_data, int* vars,int* dev_mapped_formula,int* dev_mapped_cp, int g_tn, int dbg);
static void HandleError(hipError_t err,const char *file,int line){
    if (err != hipSuccess){
      printf("%s in %s at line %d\n",hipGetErrorString(err),file,line);
      printf("ncl %d at lev %d\n",nclausel,level[0]);
      for(int i=1;i<trail[0];i++)
           printf("%d@%dr%d,",trail[i],level[abs(trail[i])],refs[abs(trail[i])]);
      printf("\n");
      exit( EXIT_FAILURE );}}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#define HANDLE_NULL( a ){if (a == NULL) { \
     printf( "Host memory failed in %s at line %d\n", __FILE__, __LINE__ ); \
     exit( EXIT_FAILURE );}}
#define IMIN(a,b) (a<b?a:b)
#define IMAX(a,b) (a>b?a:b)


void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "ERRORE CUDA: >%s<: >%s<.\n", msg, hipGetErrorString(err) );
        exit(-1);
    }                         
}



// *** ALLOCATE / DEALLOCATE

__host__ void allocate_first(){
//*** SIZE INDEPENDENT ON THE FORMULA  
  if (USEGPU>0){
    HANDLE_ERROR( hipHostAlloc( (void**)&formula,         MAX_NL * sizeof(int), hipHostMallocDefault ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_mapped_formula, MAX_NL * sizeof(int)));
    HANDLE_ERROR( hipMalloc((void**)&dev_formula,         MAX_NL * sizeof(int)));
    HANDLE_ERROR( hipHostAlloc( (void**)&clause_pointer, (MAX_NC+1) * sizeof(int), hipHostMallocDefault ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_mapped_cp,     (MAX_NC+1) * sizeof(int) ));
    HANDLE_ERROR( hipMalloc((void**)&dev_cp,             (MAX_NC+1) * sizeof(int) ));

    // mode 5
    HANDLE_ERROR( hipMalloc((void**)&dev_vars_w,          DV_SIZE_W*((CUDABLOCKS*CUDATHREADS)) * sizeof(int)));
    HANDLE_ERROR( hipMalloc((void**)&dev_v2c_w,          2*DV_SIZE_W*((CUDABLOCKS*CUDATHREADS)) * sizeof(int)));
    HANDLE_ERROR( hipMalloc((void**)&dev_w_w,          2*DV_SIZE_W*((CUDABLOCKS*CUDATHREADS)) * sizeof(int)));
    HANDLE_ERROR( hipMalloc((void**)&dev_next_w,          2*DV_SIZE_W*((CUDABLOCKS*CUDATHREADS)) * sizeof(int)));
    HANDLE_ERROR( hipMalloc((void**)&dev_unit_w,          DV_SIZE_W*((CUDABLOCKS*CUDATHREADS)) * sizeof(int)));
    HANDLE_ERROR( hipMalloc((void**)&dev_c2v_w,          2*DV_SIZE_W*((CUDABLOCKS*CUDATHREADS)) * sizeof(int)));
    HANDLE_ERROR( hipMalloc((void**)&dev_varstack_w,          2*DV_SIZE_W*((CUDABLOCKS*CUDATHREADS)) * sizeof(int)));

  }
  else{
    formula=(int*)malloc(MAX_NL * sizeof(int));
    clause_pointer=(int*)malloc((MAX_NC+1) * sizeof(int));
  }
  filtered_clause_pointer=(int*) malloc((MAX_NC+1) * sizeof(int));
  filtered_formula       =(int*) malloc(MAX_NL * sizeof(int));
}    
    
__host__ void allocate_second(){
#ifdef MYTRACE
    printf("Memory allocation (in main):\n");
    printf("    (cuda)                     phi:\t%d bytes for %d literals\n", NL * sizeof(int), NL);
    printf("    (cuda)                      cp:\t%d bytes for %d clauses\n", (NC + 1) * sizeof(int), NC);
    printf("    (cuda)               host_vars:\t%d bytes for %d vars\n", NV * sizeof(int), NV);
    printf("    (cuda)          dev_parma_vars:\t%d bytes for %d vars\n", NV * sizeof(int), NV);
    printf(" (host)                 h_mask_num:\t%d bytes for %d blocks\n", PARblocks * sizeof(int), PARblocks);
    printf(" (host)                  h_mask_id:\t%d bytes for %d blocks\n", PARblocks * sizeof(int), PARblocks);
    fflush(stdout);
#endif

//*** SIZE DEPENDING ON THE FORMULA
//INITandy
    if ( (strategia_varSelHost == JEROWANGONESIDED) || (strategia_varSelHost == DLIS)) {
	host_strat_vars_weight=(uint*)malloc((2*NV) * sizeof(uint));
	host_strat_vars_Oneocc=(int*)malloc((2*NV) * sizeof(int));
	memset(host_strat_vars_weight,((uint)0),(2*NV)*sizeof(uint));
	memset(host_strat_vars_Oneocc,((int)0),(2*NV)*sizeof(int));
    }
    if ( (strategia_varSelHost == JEROWANGTWOSIDED) || (strategia_varSelHost == DLCS) ) {
	host_strat_vars_weight=(uint*)malloc(NV * sizeof(uint));
	host_strat_vars_Oneocc=(int*)malloc(NV * sizeof(int));
	memset(host_strat_vars_weight,((uint)0),NV*sizeof(uint));
	memset(host_strat_vars_Oneocc,((int)0),NV*sizeof(int));
    }
    if ( (strategia_varSelHost == MINCLAUSE) && (strategia_maps_varSort == MAPSVARSORT_YES) ) {
	host_strat_vars_weight=(uint*)malloc(NV * sizeof(uint));
	memset(host_strat_vars_weight,((uint)0),NV*sizeof(uint));
    }
    permutator_maps_vars = (int*) malloc(NV * sizeof( int));
//FINEandy

    host_vars=(int*)malloc(NV * sizeof(int)); 

    if (USEGPU>0){
      HANDLE_ERROR( hipHostAlloc( (void**)&h_mask_data, (3* PARblocks+NV) * sizeof(int),  hipHostMallocDefault ) ); //questo non serve su solo cpu 
      HANDLE_ERROR( hipMalloc((void**)&dev_vars,              NV * sizeof( int) )) ;
      HANDLE_ERROR( hipMalloc( (void**)&dev_parma_vars,       NV * sizeof(int) ) );
      HANDLE_ERROR( hipMalloc( (void**)&mask_data,      (3* PARblocks+NV) * sizeof(int) ) );
    }
    filtered_vars = (int*) malloc(NV * sizeof( int));
    maps_vars     = (int*) malloc(NV * sizeof( int));
    level         = (int*) malloc(NV * sizeof(int));
    refs          = (int*) malloc(NV * sizeof(int));
    trail         = (int*) malloc(NV * sizeof(int));
    seen          = (int*) malloc(NV * sizeof(int));
    learnt_clause = (int*) malloc(NV * sizeof(int)); // AGO" giusto NV?
    // *** Initialization:
    if (USEGPU>0){
    HANDLE_ERROR( hipMemcpy( dev_mapped_formula, formula,    NL * sizeof(int), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_mapped_cp, clause_pointer, (NC+1) * sizeof(int), hipMemcpyHostToDevice ) );
    }
    for(int i=0;i<NV;i++){host_vars[i]=-1; level[i] = -1; refs[i] = -1; seen[i]=0;}
    trail[0] = 1; 
    level[0] = 0;
}

__host__ void deallocate() {      
  free( host_vars );  //AGO: solo queste deallochiamo?
  if (USEGPU>0){
    HANDLE_ERROR( hipFree( mask_data ));      //
    HANDLE_ERROR( hipFree(dev_vars));
    HANDLE_ERROR( hipFree(dev_formula));
    HANDLE_ERROR( hipFree(dev_cp));
    hipEventDestroy( start );
    hipEventDestroy( stop );
  }
}
    

//**************************************************
// INPUT: 
// Data Structure: the formula is stored in two vectors.
// The first stores the literals in sequence. 
// Its length is NL (0..NL-1)
// The second stores the beginning of the various clauses
// The number of clauses is NC. The vector is 0..NC
// The last (redundant) cell stores the value NL
// It also returns NV = 1+number of vars in the formula
// Remark: the formula is read twice 
//**************************************************

__host__ int simple_parsing(char c, FILE *miofile){
  int sum=c-'0';
  char t = fgetc(miofile);
  while (47 <t  && t < 58){
       sum=sum*10+t-48;
       t = fgetc(miofile);
  }
  return sum;   
}

__host__ void leggilinea(FILE *miofile){
  char t = fgetc(miofile);
  while (t != 13 && t != 10) // Look for CR or NL
       t = fgetc(miofile);
}

__host__ void load_formula(char *filename, int *NV, int *NC,int *NL){
  int i=0,j=1,max = 0,flag=0;
  char t;
  FILE *miofile;

  clause_pointer[0]=0;  
  if((miofile = fopen(filename,"r"))==NULL) {
     printf("File not found >%s< (premature exit)\n",filename); exit(-1);
  }
  else { 
    while(!feof(miofile)){
      t = fgetc(miofile);
      if (t=='c' || t=='p' || t=='%')
         leggilinea(miofile); // Skip comments
      else if (t=='-'){       
         flag=1;
         formula[i] = -simple_parsing(48,miofile);
         max = IMAX(max,-formula[i]); 
         i++;
      }      
      else if ('0' <t  && t <= '9'){
         flag=1;
         formula[i] = simple_parsing(t, miofile);
         max = IMAX(max,formula[i]); 
         i++;
      } 
      else if (t == '0' && flag==1){
         clause_pointer[j]=i; j++; flag=0;
      }   
    }            
    printf("Num clauses: %d, Num vars: %d, Num literals: %d \n",j-1,max,i);       
    *NV = max+1;   *NC = j-1;    *NL = i;
    fclose(miofile);     
  } // else  
}

//**************************************************
// OUTPUT functions: 
//**************************************************

//INITandy
void verify_solution(int *vars) {
	int cissat=0; int i,j;
	int fissat=1;
	for (i=0; i < NC; i++) {
		cissat=1;
		j=clause_pointer[i];
		while (j<clause_pointer[i+1]) {
			if ((formula[j]<0 && vars[-formula[j]]!=1)||(formula[j]>0 && vars[formula[j]]!=0)) {
				cissat=1;
				break;
			}
		j++;
		}
		if (cissat == 0) {
			printf("clausola %d violata\n",i);
			fissat = 0;
			break;
		}
	}
	printf("\nSOLUTION CHECK (0==False, 1==True, -1==Undef): %s\n\n",((fissat==1)?"***OK***":"### ALERT ### FORMULA NOT SATISFIED!!! #####"));fflush(stdout);
}


void print_option (int option) {
switch (option) {
	//case NONE: { printf("NONE"); break; }
	case MINCLAUSE: { printf("MINCLAUSE"); break; }
	case JEROWANGTWOSIDED: { printf("JEROWANGTWOSIDED"); break; }
	case JEROWANGONESIDED: { printf("JEROWANGONESIDED"); break; }
	case DLCS: { printf("DLCS"); break; }
	case DLIS: { printf("DLIS"); break; }
	default: { printf("DEFAULT"); break; }
}
}
//FINEandy

__host__ void  print_info(hipDeviceProp_t prop){
    printf("INFO: Variable reordering strategy (before calling the device): %s\n", (strategia_maps_varSort==MAPSVARSORT_YES)?"YES":"NO");fflush(stdout);
//    printf("INFO: multiproc %d\n",prop.multiProcessorCount);
//    printf("INFO: sh mem per block %d\n",(int)prop.sharedMemPerBlock);
//    printf("INFO: threads per block %d\n",prop.maxThreadsPerBlock);
    printf("INFO: kernelExecTimeoutEnabled %d\n",prop.kernelExecTimeoutEnabled);
//    printf("INFO: max tex 1d %d\n",prop.maxTexture1D);
//    printf("INFO: max tex 2d %d %d\n",prop.maxTexture2D[0],prop.maxTexture2D[1]);
//    printf("INFO: tex align %d\n",(int)prop.textureAlignment);
    printf("INFO: Block Allocation ");
    #ifdef SIMPLEBLOCK
        printf("simple\n");
    #else
        printf("with unit propagation\n");
    #endif
}

__host__ void print_instance( int * phi, int * cp, int NC){
      int i=0,j=1; 
      printf("SAT INSTANCE: \n");
      while(j <= NC){
         printf("%d ",phi[i]);
         i++;
         if(i == cp[j]){printf("\n");j++;}
      }   
      printf("*****************\n"); 
}       

__host__ void print_result(int *vars, int NV){
  printf("\n %d GPU mask prop %d GPU lower tree search, %d backtracks, %d backjumps, %d clauses learnt\n",CUDA_count,CUDA_count_lower,
	 backtracking,backjump,nclausel);
    if (vars[0]){
         for(int i=1;i<NV;i++){
           printf( "V[%d]=%d", i, vars[i]);
           if (i < NV-1) printf(", ");
           if (i % 7 == 0) printf("\n");
        }
        printf("\n");
	verify_solution(vars);
    }    
    else if  (vars[0]==-1)
         printf( "Computation ended by timeout\n");
    else printf( "No solution\n");
}    

// print_time prints a time expressed in ms 

__host__ void print_time(float time){
    //int MIN;
    //if (time>=60000){
    //      MIN = (int)time/60000;
    //      time = (time - MIN*60000)/1000;
    //      printf("Execution Time: %d m %.5f s\n",MIN,time);
    //}      
    //else
     printf("Execution Time: %.5f sec\n",time/1000);
}


//*****************************************************
//*** PROPAGATION PHASE
//*****************************************************

//INITandy
// unit propagation is implemented by mask_prop (in one of its variants).
// Given a (partial) assignment "host_vars", for each clause i, 
// mask= 0 => clause is true 
// mask=-1 => clause i is false 
// mask> 0 => clause i can be satisfied (there are unground variables)
// *****************************************************************
// return  0 if all clauses are satisfied
// return -1 if there is a false clause
// return  >0  otherwise and in this case *clauseind is the index of a
// **** clause containing *sel_var 
// **** sel_var is a pointer to the (selected) unassigned literal
// **** *minmask is a weight of the selected clause/literal
// ******************************************************************

//  BIG_EXP meno di __WORDSIZE 
// usato in JEROWANG*  (2^BIG_EXP e' il contributo max di una occorrenza di una var)
#define BIG_EXP 20

__host__ int mask_prop(int* sel_var, int* minmask, int* clauseind){
  int i,j,min=0,t,mask=0,pt,bestpt=0,bestcl=0;
  bool initial=true;
  int maybe_sat; // 2 surely sat, 1 maybe, 0 unsat
  
  //***********************************************************************
  //****** SIMPLE CASE (coherent with mask_prop_delta and PARMASKprop)
  //***********************************************************************
  // *** AGO: some little speed-up can be obtained storing pt=t instead of
  // *** pt = j (and changing all tests accordingly - basically removing formula[...]). 
  // *** Left this way to be fully consistent with mask_prop_delta
  //***********************************************************************

*clauseind=0;
*minmask=0;

switch (strategia_varSelHost) {

/*
case NONE: { // prende la prima che trova
  i=0;    
  while((mask >= 0) && (i <NC)){
    mask = 0;   
    j=clause_pointer[i];
    maybe_sat = 0;
    pt = 0; // Greater than number of vars
    while((maybe_sat < 2) && (j<clause_pointer[i+1])) {
      t=formula[j]; // t = literal in position j 
      if ((t<0 && host_vars[-t]==0)||(t>0 && host_vars[t]==1)){
	mask      = 0;
	maybe_sat = 2; 
	break;
      }  
      else if (host_vars[abs(t)] < 0) { // Literal Unknown 
	mask++; 
	if (pt == 0 || (pt > 0 && abs(t) < abs(formula[pt]))) 
	  pt=j;  // ==> Here the remark above
	j++;
	maybe_sat = 1;
      } 
      else j++;
    }  // while
    //if (j==clause_pointer[i+1] && maybe_sat==0) {mask = -1;  }// EOC: UNSAT!
    if (j==clause_pointer[i+1] && maybe_sat==0) { // EOC: UNSAT!
	    mask = -1;
	    *minmask = -1;
	    sel_var=NULL; //SERVE avere uno dei lit di clauseind?
	    *clauseind=i;
#ifdef TRACESELECT
    printf("-");fflush(stdout);
#endif
	    return(-1);
    }
    if (mask != 0) { //trovata
       min=mask; bestpt=pt; bestcl=i; break;
    }
    i++;     
  } // while principale
  //  OUTPUT 
  *sel_var  =  formula[bestpt];
  *minmask =  min;
  *clauseind=bestcl;
#ifdef TRACESELECT
	printf("%s", min==0 ? "+" : (min==1 ? "u": "N"));fflush(stdout);
#endif
  return(min);        

} // FINE STRATEGIA NONE
*/

case MINCLAUSE: { //variabile che occorre nella clausola piu' corta
  i=0;    
  if (strategia_maps_varSort == MAPSVARSORT_YES) {memset(host_strat_vars_weight,((uint)0),NV*sizeof(uint)); }
//VERSIONE ORIGINALE:
/*
  while((mask >= 0) && (i <NC)){
    mask = 0;
    j=clause_pointer[i];
    maybe_sat = 0;
    pt = 0; // Greater than number of vars
    while((maybe_sat < 2) && (j<clause_pointer[i+1])) {
      t=formula[j]; // t = literal in position j 
      if ((t<0 && host_vars[-t]==0)||(t>0 && host_vars[t]==1)){
        mask      = 0;
        maybe_sat = 2;
      }
      else if (host_vars[abs(t)] < 0) { // Literal Unknown 
        mask++;
if (strategia_maps_varSort == MAPSVARSORT_YES) {
//printf("INC host_strat_vars_weight[abs(t)] t=%d\n",t); 
(host_strat_vars_weight[abs(t)])++;}

        if (pt == 0 || (pt > 0 && abs(t) < abs(formula[pt])))
          pt=j;  // ==> Here the remark above
        j++;
        maybe_sat = 1;
      }
      else j++;
    }  // while
    if (j==clause_pointer[i+1] && maybe_sat==0) mask = -1; // EOC: UNSAT!
    if (initial) {
      if (mask != 0)
        {min=mask; bestpt=pt; initial=false; bestcl=i;}
    }
    else if (mask!=0){ // not initial
      if  (mask<min || (mask==min && abs(formula[pt]) < abs(formula[bestpt])) ||
           (mask==min && abs(formula[pt]) == abs(formula[bestpt]) && i < bestcl))
        {min=mask; bestpt=pt; bestcl=i;}
    }
    i++;
  } // while principale
*/

//VERSIONE MODIFICATA:
// /*
  while((mask >= 0) && (i <NC)){
    mask = 0;   
    j=clause_pointer[i];
    maybe_sat = 0;
    pt = 0; // Greater than number of vars
    while((maybe_sat < 2) && (j<clause_pointer[i+1])) {
      t=formula[j]; // t = literal in position j 
      if ((t<0 && host_vars[-t]==0)||(t>0 && host_vars[t]==1)){
	mask      = 0;
	maybe_sat = 2; 
	break;
      }  
      else if (host_vars[abs(t)] < 0) { // Literal Unknown 
	mask++; 
	if (strategia_maps_varSort == MAPSVARSORT_YES) {(host_strat_vars_weight[abs(t)])++;}
	if (pt == 0 || (pt > 0 && abs(t) < abs(formula[pt]))) 
	  pt=j;  // ==> Here the remark above
	j++;
	maybe_sat = 1;
      } 
      else j++;
    }  // while
    //if (j==clause_pointer[i+1] && maybe_sat==0) {mask = -1;  }// EOC: UNSAT!
    if (j==clause_pointer[i+1] && maybe_sat==0) { // EOC: UNSAT!
	    mask = -1;
	    *minmask = -1;
	    sel_var=NULL; //SERVE avere uno dei lit di clauseind?
	    *clauseind=i;
#ifdef TRACESELECT
    printf("-");fflush(stdout);
#endif
	    return(-1);
    }
    if (initial) {  // non ho ancora trovato clausole/lit
      if (mask != 0){
	min=mask; bestpt=pt; initial=false; bestcl=i;
      } 
    }       
    else if (mask!=0){ // not initial
      if  ((mask<min)  || (mask==min && abs(formula[pt]) < abs(formula[bestpt])) ||
	   (mask==min && abs(formula[pt]) == abs(formula[bestpt]) && i < bestcl))
	{min=mask; bestpt=pt; bestcl=i;}
    }
    i++;     
  } // while principale
// */

  //*** OUTPUT 
  *sel_var  =  formula[bestpt];
  *minmask =  min;
  *clauseind=bestcl;
#ifdef TRACESELECT
	printf("%s", min==0 ? "+" : (min==1 ? "u": "M"));fflush(stdout);
#endif
  return(min);        

} // FINE STRATEGIA MIN-CLAUSE



case JEROWANGTWOSIDED: {
  *minmask=0;
  i=0;    
  memset(host_strat_vars_weight,((uint)0),NV*sizeof(uint));
  //memset(host_strat_vars_Oneocc,((int)0),NV*sizeof(int)); //NONSERVE
  while (i < NC) {
    mask = 0;   
    maybe_sat = 0;
    j=clause_pointer[i];
    while (j<clause_pointer[i+1]) {
      if ((formula[j]<0 && host_vars[-formula[j]]==0)||(formula[j]>0 && host_vars[formula[j]]==1)){ //clausola sat
	mask      = 0;
	maybe_sat = 2;
	break; 
      } else if (host_vars[abs(formula[j])] < 0) { // Literal Unknown 
	mask++; 
	t=j;  //memo id del lit in formula[]
	maybe_sat = 1;
      } 
      j++;
    }  // while (lits della clausola)
    if (j==clause_pointer[i+1] && maybe_sat==0) { // EOC: UNSAT!
	    mask = -1;
	    *minmask = -1; sel_var=NULL;
	    *clauseind=i;
#ifdef TRACESELECT
	printf("-");fflush(stdout);
#endif
	    return(-1);
    }
    if (mask>1) {
      for (j=clause_pointer[i]; j<clause_pointer[i+1]; j++) {
	      if (host_vars[abs(formula[j])]<0) {
		      host_strat_vars_weight[abs(formula[j])] += (1<< IMAX(1,(BIG_EXP-mask)));
		      host_strat_vars_Oneocc[abs(formula[j])] = i*(formula[j]<0?-1:1);  // ANDY: gestione grezza!! DA MIGLIORARE (memo il segno del lit)
	      }
      } // for (lits della clausola)
    } else if (mask==1) {
	    *minmask=1; *clauseind=i; *sel_var=formula[t]; //t is idx of last Unkn Found;
#ifdef TRACESELECT
	printf("u");fflush(stdout);
#endif
	    return(1);
    }
    i++;
  } // while (i) principale (sulle clausole)

  //cerco atomo unknown con la best weight (riciclo la variabile bestpt)
  for (j=1; j<NV; j++) { //cerco il primo pesante 
    if (host_strat_vars_weight[j]>0) {
	    bestpt=j;
	    break;
    }
  } 
  if (j==NV) {
#ifdef TRACESELECT
	printf("+");fflush(stdout);
#endif
	  return(0); //nessun lit unknown e formula non unsat
  }
  j++;
  for (; j<NV; j++) { // cerco il piu' pesante
    if (host_strat_vars_weight[bestpt]<host_strat_vars_weight[j]) {
	    bestpt=j;
    } 
  } 
  *clauseind = abs(host_strat_vars_Oneocc[bestpt]);       // indice di una clausola (qualsiasi, ORA e' l'ultime trovata) che contenga select lit 
  *sel_var  =  bestpt * (host_strat_vars_Oneocc[bestpt]<0 ?-1:1); // segno della var selected in tale clausola
  //*sel_var  =  bestpt * (host_strat_vars_Oneocc[bestpt]>0 ?-1:1); // segno opposto rispetto a MINCLAUSE
  *minmask =  IMIN(INT_MAX,host_strat_vars_weight[bestpt]);
#ifdef TRACESELECT
	printf("J");fflush(stdout);
#endif
  return(*minmask); //basta return di un >1

} // FINE STRATEGIA JEROWANGTWOSIDED


case JEROWANGONESIDED: {
  *minmask=0;
  i=0;    
  memset(host_strat_vars_weight,((uint)0),(2*NV)*sizeof(uint));
  //memset(host_strat_vars_Oneocc,((int)0),(2*NV)*sizeof(int)); //NONSERVE
  while (i < NC) {
    mask = 0;   
    maybe_sat = 0;
    j=clause_pointer[i];
    while (j<clause_pointer[i+1]) {
      if ((formula[j]<0 && host_vars[-formula[j]]==0)||(formula[j]>0 && host_vars[formula[j]]==1)){ //clausola sat
	mask      = 0;
	maybe_sat = 2;
	break; 
      } else if (host_vars[abs(formula[j])] < 0) { // Literal Unknown 
	mask++; 
	t=j;  //memo id del lit in formula[]
	maybe_sat = 1;
      } 
      j++;
    }  // while (lits della clausola)
    if (j==clause_pointer[i+1] && maybe_sat==0) { // EOC: UNSAT!
	    mask = -1;
	    *minmask = -1; sel_var=NULL;
	    *clauseind=i;
#ifdef TRACESELECT
	printf("-");fflush(stdout);
#endif
	    return(-1);
    }
    if (mask>1) {
      for (j=clause_pointer[i]; j<clause_pointer[i+1]; j++) {
	      if (host_vars[abs(formula[j])]<0) {
		      if (formula[j]<0) {
			      host_strat_vars_weight[NV-formula[j]] += (1<< IMAX(1,(BIG_EXP-mask)));
			      host_strat_vars_Oneocc[NV-formula[j]] = i;  // la clausola in cui occorre
		      } else {
			      host_strat_vars_weight[formula[j]] += (1<< IMAX(1,(BIG_EXP-mask)));
			      host_strat_vars_Oneocc[formula[j]] = i;  // la clausola in cui occorre
		      }
	      }
      } // for (lits della clausola)
    } else if (mask==1) {
	    *minmask=1; *clauseind=i; *sel_var=formula[t]; //t is idx of last Unkn Found;
#ifdef TRACESELECT
	printf("u");fflush(stdout);
#endif
	    return(1);
    }
    i++;
  } // while (i) principale (sulle clausole)

  //cerco atomo unknown con la best weight (riciclo la variabile bestpt)
  for (j=1; j<(2*NV); j++) { //cerco il primo pesante 
    if (host_strat_vars_weight[j]>0) {
	    bestpt=j; break;
    }
  } 
  if (j==(2*NV)) {
#ifdef TRACESELECT
	printf("+");fflush(stdout);
#endif
	  return(0); //nessun lit unknown e formula non unsat
  }
  j++;
  for (; j<(2*NV); j++) { // cerco il piu' pesante
    if (host_strat_vars_weight[bestpt]<host_strat_vars_weight[j]) {
	    bestpt=j;
    }
  } 
  *clauseind = host_strat_vars_Oneocc[bestpt];       // indice di una clausola (qualsiasi, ORA e' l'ultima trovata) che contenga select lit 
  *sel_var  =  (bestpt < NV)? bestpt : -(bestpt-NV); 
  *minmask =  IMIN(INT_MAX,host_strat_vars_weight[bestpt]);
#ifdef TRACESELECT
	printf("J");fflush(stdout);
#endif
  return(*minmask); //basta return di un >1

} // FINE STRATEGIA JEROWANGONESIDED




case DLCS: { //Dynamic Largest Combined Sum
  *minmask=0;
  i=0;    
  memset(host_strat_vars_weight,((uint)0),NV*sizeof(uint));
  while (i < NC) { //si potrebbe fare con un solo loop su formula[]
    mask = 0;   
    maybe_sat = 0;
    j=clause_pointer[i];
    while (j<clause_pointer[i+1]) {
      if ((formula[j]<0 && host_vars[-formula[j]]==0)||(formula[j]>0 && host_vars[formula[j]]==1)){ //clausola sat
	mask      = 0;
	maybe_sat = 2;
	break; 
      } else if (host_vars[abs(formula[j])] < 0) { // Literal Unknown 
	mask++; 
	t=j;  //memo id del lit in formula[]
	maybe_sat = 1;
      } 
      j++;
    }  // while (lits della clausola)
    if (j==clause_pointer[i+1] && maybe_sat==0) { // EOC: UNSAT!
	    mask = -1;
	    *minmask = -1; sel_var=NULL;
	    *clauseind=i;
#ifdef TRACESELECT
	printf("-");fflush(stdout);
#endif
	    return(-1);
    }
    if (mask>1) {
      for (j=clause_pointer[i]; j<clause_pointer[i+1]; j++) {
	      if (host_vars[abs(formula[j])]<0) {
		      (host_strat_vars_weight[abs(formula[j])])++;
		      host_strat_vars_Oneocc[abs(formula[j])] = i*(formula[j]<0?-1:1);  // ANDY: gestione grezza!! DA MIGLIORARE (memo il segno del lit)
	      }
      } // for (lits della clausola)
    } else if (mask==1) {
	    *minmask=1; *clauseind=i; *sel_var=formula[t]; //t is idx of last Unkn Found;
#ifdef TRACESELECT
	printf("u");fflush(stdout);
#endif
	    return(1);
    }
    i++;
  } // while (i) principale (sulle clausole)

  //cerco atomo unknown con la best weight (riciclo la variabile bestpt)
  for (j=1; j<NV; j++) { //cerco il primo pesante 
    if (host_strat_vars_weight[j]>0) {
	    bestpt=j;
	    break;
    }
  } 
  if (j==NV) {
#ifdef TRACESELECT
	printf("+");fflush(stdout);
#endif
	  return(0); //nessun lit unknown e formula non unsat
  }
  j++;
  for (; j<NV; j++) { // cerco il piu' pesante
    if (host_strat_vars_weight[bestpt]<host_strat_vars_weight[j]) {
	    bestpt=j;
    } 
  } 
  *clauseind = abs(host_strat_vars_Oneocc[bestpt]);       // indice di una clausola (qualsiasi, ORA e' l'ultime trovata) che contenga select lit 
  *sel_var  =  bestpt * (host_strat_vars_Oneocc[bestpt]<0 ?-1:1); // segno della var selected in tale clausola
  *minmask =  IMIN(INT_MAX,host_strat_vars_weight[bestpt]);
#ifdef TRACESELECT
	printf("C");fflush(stdout);
#endif
  return(*minmask); //basta return di un >1

} // FINE STRATEGIA DCIS


case DLIS: { //Dynamic Largest Individual Sum
  *minmask=0;
  i=0;    
  memset(host_strat_vars_weight,((uint)0),(2*NV)*sizeof(uint));
  //memset(host_strat_vars_Oneocc,((int)0),(2*NV)*sizeof(int)); //NONSERVE
  while (i < NC) {
    mask = 0;   
    maybe_sat = 0;
    j=clause_pointer[i];
    while (j<clause_pointer[i+1]) {
      if ((formula[j]<0 && host_vars[-formula[j]]==0)||(formula[j]>0 && host_vars[formula[j]]==1)){ //clausola sat
	mask      = 0;
	maybe_sat = 2;
	break; 
      } else if (host_vars[abs(formula[j])] < 0) { // Literal Unknown 
	mask++; 
	t=j;  //memo id del lit in formula[]
	maybe_sat = 1;
      } 
      j++;
    }  // while (lits della clausola)
    if (j==clause_pointer[i+1] && maybe_sat==0) { // EOC: UNSAT!
	    mask = -1;
	    *minmask = -1; sel_var=NULL;
	    *clauseind=i;
#ifdef TRACESELECT
	printf("-");fflush(stdout);
#endif
	    return(-1);
    }
    if (mask>1) {
      for (j=clause_pointer[i]; j<clause_pointer[i+1]; j++) {
	      if (host_vars[abs(formula[j])]<0) {
		      if (formula[j]<0) {
			      (host_strat_vars_weight[NV-formula[j]])++;
			      host_strat_vars_Oneocc[NV-formula[j]] = i;  // la clausola in cui occorre
		      } else {
			      (host_strat_vars_weight[formula[j]])++;
			      host_strat_vars_Oneocc[formula[j]] = i;  // la clausola in cui occorre
		      }
	      }
      } // for (lits della clausola)
    } else if (mask==1) {
	    *minmask=1; *clauseind=i; *sel_var=formula[t]; //t is idx of last Unkn Found;
#ifdef TRACESELECT
	printf("u");fflush(stdout);
#endif
	    return(1);
    }
    i++;
  } // while (i) principale (sulle clausole)

  //cerco atomo unknown con la best weight (riciclo la variabile bestpt)
  for (j=1; j<(2*NV); j++) { //cerco il primo pesante 
    if (host_strat_vars_weight[j]>0) {
	    bestpt=j; break;
    }
  } 
  if (j==(2*NV)) {
#ifdef TRACESELECT
	printf("+");fflush(stdout);
#endif
	  return(0); //nessun lit unknown e formula non unsat
  }
  j++;
  for (; j<(2*NV); j++) { // cerco il piu' pesante
    if (host_strat_vars_weight[bestpt]<host_strat_vars_weight[j]) {
	    bestpt=j;
    }
  } 
  *clauseind = host_strat_vars_Oneocc[bestpt];       // indice di una clausola (qualsiasi, ORA e' l'ultima trovata) che contenga select lit 
  *sel_var  =  (bestpt < NV)? bestpt : -(bestpt-NV); 
  *minmask =  IMIN(INT_MAX,host_strat_vars_weight[bestpt]);
#ifdef TRACESELECT
	printf("I");fflush(stdout);
#endif
  return(*minmask); //basta return di un >1

} // FINE STRATEGIA DLIS






default: {
	 printf("\nERROR: STRATEGIA NON DEFINITA PER mask_prop.  Exiting...\n");fflush(stdout);
	 exit(-1);
 }
} // switch
} // mask_prop


//FINEandy







// ******************************************************************
// Device version without the array and with careful treatment of vars
// vars[i]<0 means still unlabeled,
// in this case, a=vars[i] is -index of deltavars for variable i

__device__ inline int mask_prop_delta(
         int * formula, int * cp,
         int * block_vars,   short int *delta_vars, 
         int NC, int * lit){
    int i=0,t,a,min=0,mask=0;
    int j;
    int pt, bestpt=0 ;
    bool initial=true;
    int maybe_sat=0; // 2 surely sat, 1 maybe, 0 unsat
    
    while((mask >= 0) && (i <NC)){
      mask = 0;   
      j=cp[i];
      maybe_sat = 0;
      pt = 0;
      while((maybe_sat < 2) && (j<cp[i+1])) {
            t=formula[j];         // Temp variables are used to reduce 
            a=block_vars[abs(t)]; // memory accesses
            if (( (t<0) && ((a==0) || ((a<0) && (delta_vars[-a]==0))))    || 
                ( (t>0) && ((a==1) || ((a<0) && (delta_vars[-a]==1)))) ) {
               mask = 0;
               maybe_sat = 2; 
            }  
            else if ((a<0) && (delta_vars[-a]== -1)){ // Literal Unknown 
               mask++; 
               if (pt == 0 || (pt > 0 && abs(t) < abs(formula[pt]))) 
                    pt=j; // CAMBIATO QUI - 22/5
               j++;
               maybe_sat = 1;
             } 
            else j++;
      }  // while
      if (j==cp[i+1] && maybe_sat==0) mask = -1; // EOC: UNSAT!
      if (initial) {
         if (mask != 0)
             {min=mask; bestpt=pt; initial=false; } 
      }       
      else if (mask!=0){ // not initial
           if  ( mask< min ||
		         (mask==min && abs(formula[pt]) < abs(formula[bestpt])))
               {min=mask; bestpt=pt;}
           }
      i++;     
   } // while principale 
   *lit = bestpt;
   return min;      
} 

//*********************************************************************
//*********************************************************************
//*********************************************************************
//*********************************************************************


__global__ void parmask_prop( int* mask_data, int* vars,int* dev_mapped_formula,int* dev_mapped_cp, int g_tn){
  int stride = blockDim.x * gridDim.x;
  int g_tid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;    
  __shared__ int s_mask_num[THREADS];
  __shared__ int s_mask_id[THREADS];   
  __shared__ int s_mask_cl[THREADS];   // memorizza la clausola che fa unit prop  

  s_mask_num[tid] = 0;  

  while(g_tid < g_tn){
    int l,v;
    short maybe_sat; // maybe_sat == 2 surely sat, 1 maybe, 0 unsat
    int iter,end;
         
    iter = dev_mapped_cp[g_tid];
    end =  dev_mapped_cp[g_tid + 1];      
	
    maybe_sat = 0;
    int num=0; // portato a registri, cosi' posso tenere s_mask per tutte le clausole
    int id=2147483647; // MAXINT <==========
    while((maybe_sat < 2) && (iter < end)) { // Per ogni lett non sat nella cl
        l = dev_mapped_formula[iter];
        v = vars[abs(l)];      
        iter++;                            // passo al prossimo lett
        if ((( l < 0 ) && ( v == 0 )) ||   // se ha segno concorde
	        (( l > 0 ) && ( v == 1 ))) {   // con la variabile ground
	         num = 0;                      // la clausola è vera
	         maybe_sat = 2;                // sicuramente sat
        }  
        else if (v == -1){ // Literal non-ground
	    // Nota. s_mask_num e s_mask_id sono shared, dunque ad accesso veloce
	         num++;          // +1 al num di lett non-ground in clausola
	         if (abs(l)<abs(id))
	             id = l;      // salvo l'indirizzo del lett 
	         maybe_sat = 1;              // maybe sat
        } 
    }  // while
    if (iter == end && maybe_sat == 0) 
      num = -1; // EndOfClause, clausola falsa.

    // aggiorna analisi clausola nello store shared
    int bestnum=s_mask_num[tid];
    int bestid=s_mask_id[tid];
    int bestcl=s_mask_cl[tid];

    // se nuovo calcolato e' piu' interessante -> memorizza!
    if( num == -1 || bestnum == -1) {
        if (num==-1 && bestnum!=-1){ // la nuova cl e' conflitto
	         s_mask_num[tid] = -1; // copia solo se il precedente non era fallito -> tengo g_tid minimo per cella tid
	         s_mask_cl[tid] = g_tid; // segno la clausola che fa fallire
        }
    }
    else if (bestnum == 0) {
        s_mask_num[tid] = num;
        s_mask_id[tid] = id;
        s_mask_cl[tid] = g_tid;
    }
    else if (num > 0 && 
	     (bestnum > num || 
         (bestnum == num && abs(bestid) > abs(id)) || 
         (bestnum == num && abs(bestid) == abs(id) && bestcl > g_tid)
	      )) {
        s_mask_num[tid] = num;
        s_mask_id[tid] = id;
        s_mask_cl[tid] = g_tid;
    }

    g_tid+=stride;
  } /// while ogni pezzo

  //*************************************************************************************
  __syncthreads();

  // DISTRIBUTED algorithm for implementing first-fail choice (and minimum)      
  short tn = blockDim.x;          
  while(tn > 1){ 
     short half = (tn >> 1); // stesso che tn = tn/2
     short tid_2 = tid + half;
     if(tid < half){
        if( s_mask_num[tid_2] == -1 || s_mask_num[tid] == -1) {
	        if ((s_mask_num[tid_2] == -1 && s_mask_num[tid]>-1 ) || 
                (s_mask_num[tid_2] == -1 && s_mask_cl[tid_2]<s_mask_cl[tid] )){ // copio se il piu' alto e' -1 e scrivo il minimo (a parita' di -1)
	           s_mask_num[tid] = -1;
	           s_mask_cl[tid] = s_mask_cl[tid_2];
	        //printf("riduzione cl %d (%d) num %d (%d)-> da %d a %d\n",s_mask_cl[tid_2],s_mask_cl[tid],s_mask_num[tid_2],s_mask_num[tid],tid_2,tid);
	        }
        }
//*** AGO GIU12: i seguenti due bodies sono uguali. Bug?
    else if (s_mask_num[tid] == 0) {
	    s_mask_num[tid] = s_mask_num[tid_2];
	    s_mask_id[tid] = s_mask_id[tid_2];
	    s_mask_cl[tid] = s_mask_cl[tid_2];
    }
    else if  (s_mask_num[tid_2] > 0 && 
		(s_mask_num[tid] > s_mask_num[tid_2] || 
          (s_mask_num[tid] == s_mask_num[tid_2] && abs(s_mask_id[tid]) > abs(s_mask_id[tid_2]))  || 
          (s_mask_num[tid] == s_mask_num[tid_2] && abs(s_mask_id[tid]) == abs(s_mask_id[tid_2]) && 
           s_mask_cl[tid] > s_mask_cl[tid_2])
		 )) {
	    s_mask_num[tid] = s_mask_num[tid_2];
	    s_mask_id[tid] = s_mask_id[tid_2];
	    s_mask_cl[tid] = s_mask_cl[tid_2];
      }
    }
    
    //*************************************************************************************
    __syncthreads();
    tn = half;
  } // while (tn>1)              

  if(tid == 0){      
    mask_data[3*blockIdx.x] = s_mask_num[0];
    mask_data[3*blockIdx.x+1] = s_mask_id[0];
    mask_data[3*blockIdx.x+2] = s_mask_cl[0];
  }
      
} // parmask_prop


//****************************************************************
//*** FORMULA SIMPLIFICATION (ATL-VEN July 12, 2011)
// simplifies a (still satisfiable) formula by 
// removing true clauses and false literals
//*****************************************************************
//INITandy
int comp(const void * a, const void * b) {
        const uint *pval1 = (uint *)(host_strat_vars_weight+*((uint *)a));
        const uint *pval2 = (uint *)(host_strat_vars_weight+*((uint *)b));
        if (*pval1 == *pval2)
                return(0);
        else
                return(*pval1 < *pval2 ? -1 : 1);
}
int compbis(const void * a, const void * b) {
        const uint *pval1 = (uint *)(host_strat_vars_weight+*((uint *)a));
        const uint *pval2 = (uint *)(host_strat_vars_weight+*((uint *)b));
        const uint *qval1 = (uint *)(host_strat_vars_weight+NV+*((uint *)a));
        const uint *qval2 = (uint *)(host_strat_vars_weight+NV+*((uint *)b));
        if ((*pval1 + *qval1) == (*pval2 + *qval2))
                return(0);
        else
                return((*pval1 + *qval1) < (*pval2 + *qval2) ? -1 : 1);
}
int pmoc(const void * a, const void * b) {
        const uint *pval1 = (uint *)(host_strat_vars_weight+*((uint *)a));
        const uint *pval2 = (uint *)(host_strat_vars_weight+*((uint *)b));
        if (*pval1 == *pval2)
                return(0);
        else
                return(*pval1 > *pval2 ? -1 : 1);
}
//FINEandy




__host__ int filter_formula(int &fv){ 

 int i,lp,cp,flp,fnc,vera;
 
 filtered_vars[0]=-1;

//INITandy
 //maps_vars[0]=0;
 memset(maps_vars,0,NV*sizeof(int));
 for (i=1; i<NV; i++) {permutator_maps_vars[i]=i;}

// se sono qui ho eseguito mask_propagation_cpu, sfrutto l'ordinamento che ha calcolato
if ((strategia_maps_varSort==MAPSVARSORT_YES) && (strategia_varSelHost != NONE )) {
    if ( (strategia_varSelHost == JEROWANGONESIDED) || (strategia_varSelHost == DLIS)) {
	qsort(permutator_maps_vars+1 , NV-1 , sizeof(int), compbis);
    } else if (strategia_varSelHost == MINCLAUSE) {
	qsort(permutator_maps_vars+1 , NV-1 , sizeof(int), pmoc);
    } else {
	qsort(permutator_maps_vars+1 , NV-1 , sizeof(int), comp);
    }
}

 //printf(".");fflush(stdout);

// Here only the unassigned (free) vars are considered. 
// maps_vars stores the correspondence new var / old var
 
 fv = 0;
 for(i=1;i<NV;i++){
    if (host_vars[i] == -1){
      fv++;
      maps_vars[i] = permutator_maps_vars[fv]; // OLD NONGROUND VARS ARE ASSIGNED TO NEW
      filtered_vars[permutator_maps_vars[fv]] = -1;
    }  
    // else maps_vars[i] = i; //(this value would be never used)
 }                         
 
/*
for(int i=1; i < NV; i++) { //if(maps_vars[i] != 0) 
	            { printf("maps_vars[%d]=%d\n",i, maps_vars[i]);   } }

for(int i=1; i < NV; i++) { //if(maps_vars[i] != 0) 
	            { printf("permutator_maps_vars[%d]=%d\n",i, permutator_maps_vars[i]);   } }
*/

// for(i=1;i<NV;i++){
//	 printf("mv[%d]=%d\n",i,maps_vars[i] );
// }
//FINEandy
 
// The formula is simplified and projected
// on the new variables.
     
 lp=0; fnc=0; 
 filtered_clause_pointer[0] = 0;
  
 for (cp=0; cp < NC; cp++){
    vera=1;
    flp = filtered_clause_pointer[fnc];
    while (lp < clause_pointer[cp+1]){
       // FIRST CASE: the literal, hence the clause C_cp is satisfied. Remove C_cp
       if ((formula[lp] > 0) && (host_vars[formula[lp]] == 1) ||
           (formula[lp] < 0) && (host_vars[-formula[lp]] == 0)){
          vera = 1;
          lp = clause_pointer[cp+1]; break; // Exit the loop
       }  // SECOND CASE: the literal is not ground. Copy it, renaming the var
       else if (host_vars[abs(formula[lp])] == -1){
          filtered_formula[flp] =  maps_vars[abs(formula[lp])];  
          if (formula[lp] <= 0)  filtered_formula[flp]=-filtered_formula[flp];
          flp++;
          lp++;
          vera=0;
       }  //THIRD CASE: the literal is false. Just skip it            
       else lp++;
   } // while
   if (!vera) { // NOTA: se e' falsa, la tolgo come se fosse vera
       fnc++;   // Non dovrebbe mai essere chiamata in tal caso.
       filtered_clause_pointer[fnc] = flp;
   }                       
  } // for  
  
  return fnc;
  
} // function filter_formula


//***************************************************************
//***** KERNEL FUNCTION.
//***** DEVICE VERSION OF DPLL
//***** Without recursion
//***** Use block/thread address to guide the search
//***** Use shared block memories
//***************************************************************

// __shared__ short int* block_varsE;

__global__ void CUDADPLL(int* formula, int* cp,  int *vars, int NC, int NV, 
              int CUDATHREADS, int LOG_BLKS, int Delta, int LOG_THRDS ){
    
  //*** If some other thread already found solution, skip the thread

  if (vars[0]<1){
   // *** SHARED ARRAY OF THE BLOCK
   __shared__  int block_vars[BV_SIZE];
   // *** LOCAL ARRAYS OF THE THREAD
   short int delta_vars[DV_SIZE], varstack[DV_SIZE][2]; 
          
   int addr=blockIdx.x,count=0,top=-1,pos,i; 
   int lit;
   bool end=false, updated=false;
   int min_mask=0;

   //****************************************************************
   // The first LOG_BLKS vars are assigned using block coordinates
   // The others are delegated to delta_vars
   //****************************************************************

     
#ifdef SIMPLEBLOCK

// *************************************************************************
// **** SIMPLE CODE. GOOD FOR DATA PARALLELISM. SOME DETERMINISM IS
// **** NOT WELL-EXPLOITED
// *************************************************************************

     if (threadIdx.x==0) {
       block_vars[0]=0;
       for(i=1;i<NV;i++){
         if (count < LOG_BLKS){ 
             block_vars[i] =  addr % 2; 
             addr=addr/2; count++;
          }         
          else { 
             block_vars[i] = top;
             top--;
          }       
       } // for i ...
     } // if ((threadIdx.x==0)...)

#else

// *************************************************************************
// **** ALTERNATIVE CODE WITH FAST UNIT PROPAGATION IN THE FIRST STAGE
// **** DATA PARALLELISM NOT POSSIBLE
// *************************************************************************
   int j,ui=0,uj=0;
   int l,v;

   if (threadIdx.x==0) {
     //**** Reset of block_vars 
     block_vars[0]=0;
     for(i=1;i<NV;i++)
        block_vars[i]=-1;
     //****
     for(i=1;i<NV;i++){
       if (count < LOG_BLKS){
            j=0; // Find the next free variable
            while((i+j < NV) && (block_vars[i+j] >= 0)) 
               j++;
            if(i+j < NV){
               block_vars[i+j] =  addr % 2;
              addr=addr/2; 
              count++;
            }
       // *** FAST UNIT PROPAGATION DIRECTLY HERE
           min_mask=1; 
           while(min_mask==1){   
             ui = 0;                 
             min_mask=0;
             while((ui < NC) && (min_mask != 1)){
                min_mask=0;  
                uj=cp[ui];
                while((min_mask < 2) && (uj < cp[ui+1])) {
                   l=formula[uj];
                   v=block_vars[abs(l)];
                   if ((( l < 0 ) && ( v == 0 )) ||   // se ha segno concorde : block -> mask=0, stop is sat!
                       (( l > 0 ) && ( v == 1 ))){
                       uj=cp[ui+1];
                       min_mask=0;
                   }
                   else if (block_vars[abs(formula[uj])] < 0) {
                      min_mask++; 
                      lit = uj;
                   } 
                   uj++;
                }  // while  min_mask < 2
                ui++;
              } // while  ui < NC  
              if (min_mask==1){
                 block_vars[ abs(formula[lit])] = (formula[lit] > 0);
              }
          }  // while(min_mask==1)
       } // if count < 
       else if(block_vars[i] == -1){   // Assign the remaining variables (if any)                    
           block_vars[i] = top;
           top--;
       }
                         
       }  // for i=1 .. NV             
     } // if ((threadIdx.x==0)...)

#endif

//*************************************************************************************
__syncthreads();


// *** Every thread resets its local part of the var array
       for(i=0;i < Delta;i++)  delta_vars[i] = -1;

// Use the (block,thread) coordinate to guide the successive 
// LOG_THRDS ND choices
// top is now a pointer to the top of the stack 
// lit points to the selected unknown literal
// pos is its sign

   addr=threadIdx.x;
   top=-1; count=0;    
   while(!end){
       if(top>=0) { delta_vars[varstack[top][0]] = varstack[top][1] % 2; }
       
       //*****************************************************
       //*** Partial substitution evaluation here:
       //*****************************************************       
       min_mask = mask_prop_delta(formula, cp, block_vars,delta_vars,NC,&lit);        
        
       if (min_mask == 0) {
          end=true; break;  // logically superfluous "break", but it speeds it up...
       } else if (min_mask > 0) { // min_mask > 0
          pos = (formula[lit] > 0); // sign of the unknown literal
          top++;  
          varstack[top][0]=-block_vars[abs(formula[lit])];
          if (min_mask == 1) // There is ONE non-ground literal: determinism 
               varstack[top][1]=pos;
          else if (count < LOG_THRDS){ // Set the variable using thread coords   
              varstack[top][1] = addr % 2; addr>>=1;
              count++;
              }         
          else // min_mask > 1 && count >= LOG_THRDS
             varstack[top][1]=2+pos; // Assign a backtrackable value
      } // else  min_mask > 0  
      
      else {   // if (min_mask < 0) *** Failure: Backtracking  
    while((top>=0) && (!updated)){
             pos=varstack[top][1]; // pos is used to avoid 2 addressing to varstack[top][1]
             if (pos > 1){   // 3-> try 0, 2-> try 1 
                 varstack[top][1] = 3-pos;   
                 updated=true;  
             } else {
                delta_vars[varstack[top][0]] = -1; // Restore unknown status
                top--;   // 1 -> stop, 0-> stop
             } 
          }
          if (top <0) {end = true; break;} else updated=false;
       } // else
    } // while

//*************************************************************************************
__syncthreads();
    
   if (vars[0]<1 && (min_mask==0)){ // *** A solution: export it on vars
     //printf("yes\n");
     vars[0]=1;
     for(pos=1; pos<NV; pos++) 
         if (block_vars[pos] >= 0)
            vars[pos] = block_vars[pos];
         else 
            vars[pos] = delta_vars[-block_vars[pos]]; 
   } 
 } // if (vars[0]...
} 


__device__ void insert_gpu(int c, int v, int p_W, int zerone, int* V2C, 
			   int* W, int* NEXT, int* C2V){
     int ptemp = V2C[v];
     V2C[v]         = p_W;
     W[p_W]         = c;
     NEXT[p_W]      = ptemp; 
     C2V[2*c+zerone] = v;
}     

__device__  int  unit_propagationw(int* U, int* UNIT, int* V2C, 
					 int* W, int* NEXT, int* C2V,
					 int* top, int* varstack,
					 int* nextselected,
					 int* host_vars,
					 int* clause_pointer,
					 int* formula,
					 int NV,
					 int dbg
					 ){
   
   int n_p,temp,c,v, lit,previous;
  
   // printf("#### ENTERED IN UNIT PROPAGATION\n");

   *nextselected = 0;
   while(*U >= 0){
      // *** v is the current (deterministic) variable

      v = abs(UNIT[*U]);
      if (dbg) printf("unit: v %d, hv %d\n",UNIT[*U],host_vars[v]);
      if (host_vars[v] < 0){
          host_vars[v] = (UNIT[*U] > 0);
          (*top)++;
          varstack[*top*2+0] = UNIT[*U]; //*** signed literal
          varstack[*top*2+1] = 2 ; //*** 2 -> not backtrackable 
	  if (dbg)
	    printf("CASE 4 - UP var[%d]=%d\n",abs(UNIT[*U]),(UNIT[*U]/abs(UNIT[*U])+1)/2);
      }// assumo che se gia' presente non e' in constrasto
      (*U)--; //*** v removed from the unit list.

      n_p = V2C[v];
      if (dbg) printf("v:%d\n",n_p);
      while (n_p > -1){ //*** Visit the v list (starting from V2C[v])       
         c = W[n_p];
         lit = clause_pointer[c];
         //*** It is not sufficient to check only the "v" based  literal!
         //*** Could be sped up with a further (backtrackable) array
         temp=1;
         while(temp && (lit < clause_pointer[c+1]))
             if ( (host_vars[abs(formula[lit])] >= 0) && ((formula[lit] > 0) == host_vars[abs(formula[lit])]))
                 temp=0;
             else lit++;
         //*** check if v satisfies the clause
         if (!temp) {
            // printf(">>> Clause %d satisfied by current assignment\n",c);
            previous = n_p;
            n_p  = NEXT[previous];
        }   
        else { //*** clause NOT satisfied by the v assignment 
               //*** seek for one unassigned literal (restart from beginning of the clause)
             // printf(">>> Clause %d **NOT** satisfied by current assignment\n",c);
             lit = clause_pointer[c];
             while((lit < clause_pointer[c+1]) && (host_vars[abs(formula[lit])] >= 0))  lit++;
             // *** Clause c is false: backtracking needed.
             if (lit == clause_pointer[c+1]){ 
	       // printf(">>> Exit due to failure\n");
                 *U = -1; // Empty the Unit list
                 return 0;
              }      
             //*** We know that formula[lit] is satisfiable. We know abs(formula[lit]) is not v   
             // *** if the first non ground literal found is already watching c, seek another
             if (abs(formula[lit]) == C2V[2*c+0] || abs(formula[lit]) == C2V[2*c+1]){
                 temp=formula[lit];
                 lit++;
                 while((lit < clause_pointer[c+1]) && (host_vars[abs(formula[lit])] >= 0)) lit++;
                // *** if there is not another:  UNIT CASE: add it to unit and update pointers
                if (lit == clause_pointer[c+1]){  
                    if (dbg) printf("Aggiungo %d a UNIT\n", temp);
                    (*U)++;
                    UNIT[*U] = temp;
                    previous = n_p;
                    n_p = NEXT[previous];
                 }                     
              } // if the first literal is watching   
              
              if (lit < clause_pointer[c+1]){ //*** Update in the non unit case
                  // **** SELECTION STRATEGY: the LAST found non ground watched literal different 
                     *nextselected = formula[lit]; // from v in the LAST non satisfied clause visited 
                     //printf("SELECTED literal %d\n",formula[lit]);
                     //*** LISTS REARRANGEMENTS
                    if (C2V[2*c+0]==v) C2V[2*c+0] = abs(formula[lit]);
                       else  C2V[2*c+1] = abs(formula[lit]);
                    temp = V2C[abs(formula[lit])];
                    if (n_p == V2C[v]){  //*** We are at the beginning of the list 
                       V2C[v] = NEXT[n_p];
                       NEXT[n_p] = temp;
                       V2C[abs(formula[lit])] = n_p;
                       n_p = V2C[v];
                    }     
                    else {
                       NEXT[previous] = NEXT[n_p];                             
                       V2C[abs(formula[lit])] = n_p;
                       NEXT[n_p] = temp;
                       n_p = NEXT[previous];
                    }
              } // Update in the non unit casa      
          } // end if clause not satisfied by "v" assignment
      } // visit the v list
    }  // while *U      

    return 1;
}        

__device__ int first_seedw(int* host_vars, int* V2C, int NV){
    int i=1,temp=1;
    
     while(temp)
          if( (i < NV) && (host_vars[i] < 0) && (V2C[i] >= 0) )
                temp = 0;
          else i++;       
     if (i < NV) 
         return i; //*** Found a non ground literals pointing
     else {
         i=1;
         temp=1;    
         while(temp)
              if( (i < NV) && (host_vars[i] < 0) )
                temp = 0;
               else i++;
         if (i < NV)  
               return i; //*** Found a non ground literal not pointing
         else return 0;
    } 
} 


///////// versione con watched literals
__global__ void CUDADPLLW(int* formula, int* clause_pointer,  int *vars, int NC, int NV, 
			  int CUDATHREADS, int LOG_BLKS, int LOG_THRDS,
			  int* host_vars_g,
			  int* V2C_g,
			  int* W_g, int* NEXT_g, int* UNIT_g, int* C2V_g, int* varstack_g){
    
  // ale: nota: qui vars scritto solo alla fine in caso di successo
  // uso host_vars come variabili locali al thread

  //printf("BL %d: TH %d: HI: var0 %d\n",blockIdx.x,threadIdx.x,vars[0]);
  int dbg=0;
  //if (threadIdx.x==0 && blockIdx.x==0) dbg=1;


//cuda-memcheck ./a.out 5 1 3 17 0 0 0 0 INS-1/02_4queens.cnf

  if (vars[0]<1){
      
      int* host_vars=&host_vars_g[DV_SIZE_W*(blockIdx.x*blockDim.x+threadIdx.x)];
      int* V2C=&V2C_g[2*DV_SIZE_W*(blockIdx.x*blockDim.x+threadIdx.x)];
      int* W=&W_g[2*DV_SIZE_W*(blockIdx.x*blockDim.x+threadIdx.x)];
      int* NEXT=&NEXT_g[2*DV_SIZE_W*(blockIdx.x*blockDim.x+threadIdx.x)];
      int* UNIT=&UNIT_g[DV_SIZE_W*(blockIdx.x*blockDim.x+threadIdx.x)];
      int* C2V=&C2V_g[2*DV_SIZE_W*(blockIdx.x*blockDim.x+threadIdx.x)];
      int* varstack=&varstack_g[2*DV_SIZE_W*(blockIdx.x*blockDim.x+threadIdx.x)];

  int i, U,sat,top, proceed, SEL=0, temp;

    
//*** RESET OF THE WATCHED STRUCTURE:

  for(i=0;i<2*NC;i++){ W[i]   = -1;  NEXT[i] = -1; }
  for(i=0;i<NC;i++){ C2V[2*i+0] = 0;  C2V[2*i+1] = 0; }
  for(i=0;i<NV;i++)  { V2C[i] = -1;  UNIT[i] = 0;  }
  U=-1;  top=-1;  sat=1; SEL=0;
  
//*** INITIALIZATION OF WATCHED MATRIX

  for(i=0;i<NC;i++){
    //if (blockIdx.x==0 && threadIdx.x==0)
      //printf("----%d------\n",i);
    //  for(i=0;i<NC;i++){
      //*** FIRST LITERAL WATCHED
    //printf("%d %d\n",i,clause_pointer[i]);
      int v=abs(formula[clause_pointer[i]]);
      int ptemp = V2C[v];
      //    printf("%d\n",v);
      V2C[v]     = 2*i;
      W[2*i]     = i;
      C2V[2*i+0] = v;
      NEXT[2*i]=ptemp;
      //*** CHECK THAT THERE ARE AT LEAST TWO LITERALS IN THE CLAUSE
      if (clause_pointer[i+1] > clause_pointer[i] + 1){
      //*** SECOND LITERAL WATCHED (IF ANY)
	 int v=abs(formula[clause_pointer[i]+1]);
	 int ptemp = V2C[v];
	 V2C[v]     = 2*i+1;
	 W[2*i+1]   = i;
	 C2V[2*i+1] = v;
	 NEXT[2*i+1]=ptemp;      
      } else { //*** READY TO APPLY UNIT PROPAGATION FOR THIS LITERAL (no repetition control!!! - this might cause error)
         U++;
         UNIT[U] = formula[clause_pointer[i]]; // Positive or negative
      }
      // printf("----%d------\n",i);

   }   
  

  if (1==0 && dbg){
   printf("**** DATA STRUCTURE *****************************************\n");  
   for(i=0;i<NC;i++)
     printf("Watched clausola %d = %d, %d\n", i, C2V[2*i+0], C2V[2*i+1]);
   for(i=0;i<2*NC;i++)
     printf("Linked structure i=%d, (%d,%d) \n", i, W[i], NEXT[i]);
   for(i=1;i<NV;i++)
     printf("Var pointer V2C[%d] = %d\n", i, V2C[i] );
   printf("**********************************************************\n");  
  }


  ///////// assegna var su stack, sia blocco che di thread
  host_vars[0]=0;
  for (int i=1;i<NV;i++)
    host_vars[i]=-1;



  int addr=(blockIdx.x<<LOG_BLKS)+threadIdx.x;
  int fixed=LOG_BLKS+LOG_THRDS;

  if (dbg) printf("add %d, %d %d\n",addr,blockIdx.x,threadIdx.x);
  //if (blockIdx.x==0) 
  //printf("bl %d th %d add %d\n",blockIdx.x,threadIdx.x,addr);




        


      int retv=-1; 

      if (retv<0){ // non trivial result
	// Heuristic first seed, if not set by unit_propagation
      
	if (SEL == 0) SEL = first_seedw(host_vars,V2C,NV); 

	// *** Initialize the stack with a backtrackable var      
	top++;
	varstack[2*top+0]= SEL;   // E.g. v satisfied by 1 -> -1
	varstack[2*top+1]= 1-addr%2; // cosi' e' backtrack, ma solo 1 val
	addr/=2;
	fixed--;
                       
	//****************************************************************************
	//*** MAIN LOOP
	//****************************************************************************
      

	while((top >=0) && (top < NV-1)){
	  proceed = 0;
	  while((top >= 0) && !proceed){  
	    if (varstack[2*top+1] > 1){ // *** no longer  backtrackable
	    if (dbg) printf("CASE 1 - pop of var[%d]=%d\n",abs(varstack[2*top+0]),host_vars[abs(varstack[2*top+0])]);
	    host_vars[abs(varstack[2*top+0])] = -1;
	    varstack[2*top+1] = -1;
	    top--;// Restore unknown status              
	  } else if (varstack[2*top+1] < 0) {// Never tried value. Try and update
	    if (dbg)  printf("CASE 2 - 1st attempt for var[%d]-\n",abs(varstack[2*top+0]));
	    proceed = 1;
	    host_vars[abs(varstack[2*top+0])] = 2+varstack[2*top+1]; 
	    varstack[2*top+1] = host_vars[abs(varstack[2*top+0])];
	  } else if (varstack[2*top+1] < 2) {// Last chance
	    if (dbg)  printf("CASE 3 - 2nd attempt  for var[%d]=%d\n",abs(varstack[2*top+0]),1-varstack[2*top+1]);
	    proceed = 1;
	    host_vars[abs(varstack[2*top+0])] = 1-varstack[2*top+1];
	    varstack[2*top+1] = 2;
	  } 
        }// *** while (! proceed)
if (dbg) printf("top %d\n",top);             
	if (top >= 0){
	  U++;
	  UNIT[U]= varstack[2*top+0];
	  if (dbg) printf("add unit %d %d\n",U,varstack[2*top+0]);
	  //*** CALL TO UNIT PROPAGATION  
	  sat =
	  unit_propagationw(&U,UNIT,V2C,W,NEXT,C2V,&top,varstack,&SEL,host_vars,clause_pointer,formula,NV,dbg);

   if (1==1 && dbg){
    printf("EXITED U.P. with U=%d, SEL=%d, top=%d, sat=%d top=%d NV=%d\n", U, SEL, top, sat,top, NV);
    if (1==1){
      for(i=1; i <NV; i++) 
	printf("v[%d]=%d",i,host_vars[i]);
      printf("\n");
    }
   }

	    if (sat && (top < NV -1) && !SEL){
	      //*** SEL becomes the next entry for U.P.
	      i=1; temp=1;
	      while(temp && (i < NV))
		if( (host_vars[i] < 0) )  
		  temp = 0;
		else i++;
	      SEL=i;
	    } 
              
	    if (sat && SEL)  {
	      top++;
	      varstack[2*top+0]= SEL;   // E.g. v satisfied by 1 -> -1
	      if (fixed>0) // ancora preassegnate
	      {
	      if (dbg) printf("force %d %d\n",SEL,addr%2);
	      varstack[2*top+1]= 1-addr%2; // cosi' e' backtrack, ma
	      addr/=2;
	      //solo 1 val, inverto il valore perche' l'assegnamento
	      // lo rigira!!	
	      
	      fixed--;	      
	      }else{
	       varstack[2*top+1]=(SEL >0) -2;     
	      }
	    }
	  
	} // if (top>=0)      
      } // *** main loop     

      retv=sat;
      }

      /// il thread ha finito
      __syncthreads();
      //if (dbg) printf("retv %d vars %d\n",retv,vars[0]);
      if (retv==1){ // *** A solution: export it on vars (potential conflict in writing solution)
	//printf("yes\n");
	vars[0]=1;
	for(int pos=1; pos<NV; pos++){
            vars[pos] = host_vars[pos];
	    if (dbg) printf("(%d %d) %d=%d\n",blockIdx.x,threadIdx.x,pos,host_vars[pos]);
	}

      }
  }
} 


//**************************************************************************
//*********    CUDA_caller                         *************************
//**************************************************************************

__host__ int CUDA_caller(){

   int unk_clauses;
   
   CUDA_count_lower++;                       
    
   // *** filter_formula call - global variables are used
   int FV;
    unk_clauses= filter_formula(FV);
    
//   DEBUG PRINT
//       printf("Called GPU with %d free vars\n",FV);
//       printf("Before: V=%d, C=%d, L=%d, After: V=%d, C=%d, L=%d\n",
//              NV-1,NC,clause_pointer[NC],FV,unk_clauses,filtered_clause_pointer[unk_clauses]);

/*
    printf("In CUDA_caller: VarsMap:\n");
            // printf("(var[%d] -> %d )\n",maps_vars[0],0);   
    for(int i=1; i < NV; i++) {
	    //if(maps_vars[i] != 0) 
	    {
	    //if(maps_vars[i] < i) 
             //printf("var[%d] -> %d \t((maps_vars[i] < i)==%d)\n",maps_vars[i],i,(maps_vars[i] < i));   
	     printf("maps_vars[%d]->%d\n",i,maps_vars[i]);   
	    }
    }
    printf("\n");fflush(stdout);

    print_instance(filtered_formula,filtered_clause_pointer,unk_clauses);  
    
    for(int i=1; i < NV; i++) 
             printf("host_vars[%d]=%d\n",i,host_vars[i]);   
    printf("\n");
*/

    //printf("unk_clau %d nl %d, fv %d\n",unk_clauses,filtered_clause_pointer[unk_clauses],FV);
    //print_instance(filtered_formula,filtered_clause_pointer,unk_clauses);  
        
    // copy the value of filtered vars, formula, and clause pointers 
    // to device global variables

    HANDLE_ERROR(hipMemcpy(dev_vars, filtered_vars, (1+FV)*sizeof( int), hipMemcpyHostToDevice ));
    HANDLE_ERROR(hipMemcpy(dev_formula, filtered_formula, filtered_clause_pointer[unk_clauses]*sizeof( int), hipMemcpyHostToDevice ));
    HANDLE_ERROR(hipMemcpy(dev_cp, filtered_clause_pointer, (unk_clauses+1)*sizeof(int), hipMemcpyHostToDevice ));
    
      	hipDeviceSynchronize();
		checkCUDAError("allocazione dev_cp execution");
#ifdef MYTRACE
  printf("\nMore memory allocation (in CUDA_caller):\n");
  printf("    (cuda)             dev_vars:\t%d bytes for %d vars\n", (1+FV)* sizeof(int), FV);
  printf("    (cuda)          dev_formula:\t%d bytes for %d vars\n", filtered_clause_pointer[unk_clauses]* sizeof(int), filtered_clause_pointer[unk_clauses]);
  printf("    (cuda)               dev_cp:\t%d bytes for %d clauses\n", (unk_clauses+1) * sizeof(int), (unk_clauses+1));
  printf(" (host)           filtered_vars:\t%d bytes for %d vars\n", (1+FV) * sizeof(int), FV);
  printf(" (host)               maps_vars:\t%d bytes for %d vars\n", NV * sizeof(int), NV);
  printf(" (host)        filtered_formula:\t%d bytes for %d vars\n", clause_pointer[NC] * sizeof(int), clause_pointer[NC]);
  printf(" (host) filtered_clause_pointer:\t%d bytes for %d clauses\n", (NC+1) * sizeof(int), NC);
  printf("CUDA_caller calling CUDADPPL3: blocks = %d  threads per block  = %d\n",CUDABLOCKS,CUDATHREADS);
  printf("          (shared) block_vars:\t%d bytes for %d vars. Total of %d bytes per block\n", MaxV * sizeof(int), MaxV, MaxV * sizeof(int));
  printf("          (thread) delta_vars:\t%d bytes for %d vars. Total of %d bytes per block\n", Delta * sizeof(int), Delta, CUDATHREADS*Delta * sizeof(int));
  printf("            (thread) varstack:\t%d bytes for %d vals. Total of %d bytes per block\n", 2*Delta * sizeof(int), 2*Delta, CUDATHREADS*2*Delta * sizeof(int));
  fflush(stdout);
#endif

    // ***************   DEVICE COMPUTATION: *************************
//    printf("Delta %d (Check if it is less than  %d)\n",Delta,DV_SIZE);
    CUDADPLL<<<CUDABLOCKS,CUDATHREADS>>>(dev_formula,dev_cp,dev_vars,unk_clauses,1+FV,CUDATHREADS,LOG_BLKS,Delta,LOG_THRDS); 

      	hipDeviceSynchronize();
		checkCUDAError("kernel CUDADPLL execution");
    // ***************   Results' analysis   *************************
    // Only the first value (a flag) is copied back
    HANDLE_ERROR(hipMemcpy(host_vars,dev_vars, sizeof( int), hipMemcpyDeviceToHost));
    //  If there is no solution, skip, otherwise copy the whole
    //  assignment back from DEVICE to HOST
    //  Then assembly the solution from vars and filtered_vars
    
    if (host_vars[0]==1){ 
      HANDLE_ERROR(hipMemcpy(filtered_vars,dev_vars, (1+FV)*sizeof( int), hipMemcpyDeviceToHost));
       for(int i = 1; i < NV; i++)  
          if (host_vars[i] < 0)          
             host_vars[i] = filtered_vars[maps_vars[i]];              
    }  
    
   
   // In realta' le prox quattro potrebbero anche essere globali o statiche,
   // comunque grandi (stanno su host) e si  evita  di riallocare
   
    return (host_vars[0]==1)*2; // 2 if good, 0 if bad
}


__host__ int CUDA_caller_watched(){

   int unk_clauses;
   
   CUDA_count_lower++;                       
    
   // *** filter_formula call - global variables are used
   int FV;
    unk_clauses= filter_formula(FV);
   
    //printf("Called GPU with %d free vars\n",FV);

//   DEBUG PRINT
       if (1==0){

       printf("Before: V=%d, C=%d, L=%d, After: V=%d, C=%d, L=%d\n",
              NV-1,NC,clause_pointer[NC],FV,unk_clauses,filtered_clause_pointer[unk_clauses]);

	   printf("Vars Mapping:\n");
	   for(int i=1; i < NV; i++) if(maps_vars[i] < i)
	   printf("var[%d] -> %d \t",maps_vars[i],i);   
	   printf("\n");

	 print_instance(filtered_formula,filtered_clause_pointer,unk_clauses);  
    
	 for(int i=1; i < NV; i++) 
	   printf("var[%d]=%d, ",i,host_vars[i]);   
	 printf("\n");

	 /*	 for(int i=1; i < FV+1; i++) 
		 printf("var[%d]=%d, ",i,filtered_vars[i]);   
		 printf("\n");
	 */
       }

       //printf("unk_clau %d nl %d, fv %d\n",unk_clauses,filtered_clause_pointer[unk_clauses],FV);
    //print_instance(filtered_formula,filtered_clause_pointer,unk_clauses);  
        
    // copy the value of filtered vars, formula, and clause pointers 
    // to device global variables

    HANDLE_ERROR(hipMemcpy(dev_vars, filtered_vars, (1+FV)*sizeof( int), hipMemcpyHostToDevice ));
    HANDLE_ERROR(hipMemcpy(dev_formula, filtered_formula, filtered_clause_pointer[unk_clauses]*sizeof( int), hipMemcpyHostToDevice ));
    HANDLE_ERROR(hipMemcpy(dev_cp, filtered_clause_pointer, (unk_clauses+1)*sizeof(int), hipMemcpyHostToDevice ));
    
#ifdef MYTRACE
  printf("\nMore memory allocation (in CUDA_caller):\n");
  printf("    (cuda)             dev_vars:\t%d bytes for %d vars\n", (1+FV)* sizeof(int), FV);
  printf("    (cuda)          dev_formula:\t%d bytes for %d vars\n", filtered_clause_pointer[unk_clauses]* sizeof(int), filtered_clause_pointer[unk_clauses]);
  printf("    (cuda)               dev_cp:\t%d bytes for %d clauses\n", (unk_clauses+1) * sizeof(int), (unk_clauses+1));
  printf(" (host)           filtered_vars:\t%d bytes for %d vars\n", (1+FV) * sizeof(int), FV);
  printf(" (host)               maps_vars:\t%d bytes for %d vars\n", NV * sizeof(int), NV);
  printf(" (host)        filtered_formula:\t%d bytes for %d vars\n", clause_pointer[NC] * sizeof(int), clause_pointer[NC]);
  printf(" (host) filtered_clause_pointer:\t%d bytes for %d clauses\n", (NC+1) * sizeof(int), NC);
  printf("CUDA_caller calling CUDADPPL3: blocks = %d  threads per block  = %d\n",CUDABLOCKS,CUDATHREADS);
  printf("          (shared) block_vars:\t%d bytes for %d vars. Total of %d bytes per block\n", MaxV * sizeof(int), MaxV, MaxV * sizeof(int));
  printf("          (thread) delta_vars:\t%d bytes for %d vars. Total of %d bytes per block\n", Delta * sizeof(int), Delta, CUDATHREADS*Delta * sizeof(int));
  printf("            (thread) varstack:\t%d bytes for %d vals. Total of %d bytes per block\n", 2*Delta * sizeof(int), 2*Delta, CUDATHREADS*2*Delta * sizeof(int));
  fflush(stdout);
#endif

    // ***************   DEVICE COMPUTATION: *************************
  if (unk_clauses>DV_SIZE_W ||FV>DV_SIZE_W){
  printf("CHECK: have %d clauses, < %d\n",unk_clauses,DV_SIZE_W);
  printf("CHECK: have %d vars, < %d\n",FV,DV_SIZE_W);
  exit(-1);
  }

  if (unk_clauses==0) return 2; //ALE: aggiunto per caso in cui filter
     		      	     	//formula toglie tutto

  int myCUDABLOCKS=CUDABLOCKS;
  int myCUDATHREADS=CUDATHREADS;
  if (FV<LOG_THRDS){
    myCUDABLOCKS=1;      
    myCUDATHREADS=(1<<FV); 
  }
  else if (FV<LOG_THRDS+LOG_BLKS){
    myCUDABLOCKS=(1<<FV-LOG_THRDS);
  }
  //printf("FV %d bl %d th %d\n",FV,myCUDABLOCKS,myCUDATHREADS);
  CUDADPLLW<<<myCUDABLOCKS,myCUDATHREADS>>>(dev_formula,dev_cp,dev_vars,unk_clauses,1+FV,myCUDATHREADS,LOG_BLKS,LOG_THRDS,
					    dev_vars_w,dev_v2c_w,dev_w_w,dev_next_w,dev_unit_w,dev_c2v_w,dev_varstack_w);
 
  HANDLE_ERROR(hipDeviceSynchronize());

  // ***************   Results' analysis   *************************
  // Only the first value (a flag) is copied back
  HANDLE_ERROR(hipMemcpy(host_vars,dev_vars, sizeof( int), hipMemcpyDeviceToHost));
  //  If there is no solution, skip, otherwise copy the whole
  //  assignment back from DEVICE to HOST
  //  Then assembly the solution from vars and filtered_vars
    
  if (host_vars[0]==1){ 
    HANDLE_ERROR(hipMemcpy(filtered_vars,dev_vars, (1+FV)*sizeof( int), hipMemcpyDeviceToHost));
    for(int i = 1; i < NV; i++)  
      if (host_vars[i] < 0)          
	host_vars[i] = filtered_vars[maps_vars[i]];              
  }  
   
  // In realta' le prox quattro potrebbero anche essere globali o statiche,
   // comunque grandi (stanno su host) e si  evita  di riallocare
  //printf("hv %d\n",host_vars[0]);
    return (host_vars[0]==1)*2; // 2 if good, 0 if bad
}


// **************************************************************************************************
// **************************************************************************************************
// *******************     clause learning **********************************************************
// **************************************************************************************************
// **************************************************************************************************

__host__ void clause_learning() {
  // sets jump and level_to_jump

	// inizializzazioni
	int i, c=0, p=0, q, var_p=0, var_q, clause, lp=0, btlevel=0, tptr=trail[0],
		verbose=0;

	if (verbose>1) {
		printf("Current level:\t\t%d\n",level[0]);
		printf("Conflict clause:\t%d\n",refs[0]);
		int i;
		printf("trail[%d] = [",trail[0]);
		for(i=1;i<trail[0]-1;i++)
			printf("%d@%d,",trail[i],level[abs(trail[i])]);
		printf("%d@%d]\n",trail[i],level[abs(trail[i])]);
	}
	//for(int u=0;u<NV;u++)
	//	if(host_vars[u]!=-1)
	//		printf("refs[%d] = %d\n",u,refs[u]);

	// *** Trovo first-UIP e creo la clausola

	//printf("\ntrail:\n");
	//for(int b=1;b<trail[0];b++)
	//	printf("%-5d @ %-5d r %-5d\n",trail[b],level[abs(trail[b])],refs[abs(trail[b])]);
	//printf("\n\n");

	memset(seen,0,NV*sizeof(int));
	do {
	  if (p==0) {				// inizio con la conflict clause 
	    clause = refs[0];	// conflict clause
	  } else
	    clause = refs[var_p];	// reason clause
	  if (verbose>1) {
	    printf("p:\t%d\t",p);
	    printf("var_p:\t%d\t",var_p);
	    printf("clause:\t%d\t",clause);
	    printf("c_start:\t%d\t",clause_pointer[clause]);
	    printf("end: %d\n",clause_pointer[clause+1]);
	    printf("reason (clause %d) is:\t",clause);
	    for(i=clause_pointer[clause];i<clause_pointer[clause+1];i++) {
	      printf("%d\t",formula[i]);
	    }
	    printf("\n");
	  }
	  //printf("clause %-6d cp[clause] %-10 clause_pointer[clause+1] %-10d var_p %-6d refs[var_p] %-10d\n",clause,clause_pointer[clause],clause_pointer[clause+1],var_p,refs[var_p]);
	  for(i=clause_pointer[clause];i<clause_pointer[clause+1];i++) {
	    var_q = abs(formula[i]);
	    q = -formula[i];
	    if ((p==0) || (var_q!=var_p)) { // se sono al primo giro (p==0) devo processare tutta la clausola, altrimenti skippo la unit propagation
	      if (verbose>1) {
		printf("q:%d\n",q);
		printf("lp at %d, q is %d at level %d, ref %d\n",lp,q,level[var_q],refs[var_q]);
	      }
	      if (!seen[var_q]) {
		if (verbose>1) printf("q never seen! ");
		seen[var_q] = 1;
		if (level[var_q] == level[0]) { // (q == decision level corrente)
		  c++;
		  if (verbose>1) printf("and of same current level (%d==%d) so c++.\n",level[var_q],level[0]);
		}
		else if (level[var_q] > 0) { // se q è di un decision level inferiore, ma non il root level (al momento il root level è inutilizzato)

		  learnt_clause[lp++] = -q;		    

		  // calcolo level to backtrack, il massimo della learnt_clause, escluso il first-uip che verrà aggiunto alla fine
		  btlevel = max(btlevel,level[var_q]);
		  if (verbose>1) {
		    printf("and of minor level (%d<%d) so add %d.\n",level[var_q],level[0],-trail[i]);
		    if (lp>0) {
		      printf("current learnt clause is: ");
		      int j;
		      for(j=0;j<lp-1;j++)
			printf("%d,",learnt_clause[j]);
		      printf("%d.\n",learnt_clause[j]);
		    }
		  }
		}
	      } else if (verbose>1) printf("q is seen\n");
	    }
	  }
	  do {
	    p = trail[--tptr];
	    var_p = abs(p);
	    if (verbose>1) {
	      printf("c:%d p%d\n",c,p);
	    }
	    if (verbose>1) {
	      printf("scan p=%d\ttptr=%d\tseen[var_p]=%d\n",p,tptr,seen[var_p]);
	    }	
	  }
	  while (!seen[var_p]);
	  c--;
	  //seen[var_p] = 0;
	  if (verbose>1) printf("c--, process var %d, c:%d, ref %d val %d\n",var_p,c,refs[var_p],host_vars[var_p]);	
	}
	while (c>0);

	//ALE: 8 giu commentata per avere backjumping
	//btlevel = max(btlevel,level[abs(p)]-1); // in caso di solo UIP senza reasons, comunque devo annullare la decisione del livello corrente (dato che poi sommo 1, qui lo sottraggo)

	// in p ora ho il first-UIP, ci va il suo negato nella learnt clause
	learnt_clause[lp++] = -p;

	// controllo inserimento clausole doppie: todo

	// debug
	if (verbose>0) {
	  printf("Add First-UIP: %d lev %d\nLearn clause %d: ",-p,level[abs(p)],NC);
	  //		printf("* LEARNT CLAUSE: ");
		for(i=0;i<lp-1;i++) {
			//printf("%d ",learnt_clause[i]); // stampa per .cnf
		  //			printf("%d@%d, ",learnt_clause[i],level[abs(learnt_clause[i])]);
		  printf("%d ",learnt_clause[i]);
		}
		//printf("%d 0\n",learnt_clause[i]);// stampa per .cnf
		//		printf("%d@%d.\n",learnt_clause[i],level[abs(learnt_clause[i])]);
		printf("%d 0\n",learnt_clause[i]);
	}
	
	// *** aggiungo la clausola imparata
	  if (//lp <10 &&
	    ((NL+lp)<=MAX_NL) && (NC+1<=MAX_NC)) { // se ho abbastanza memoria... (non è detto che debba fare forgetting!)
	  // mi arriva:	learnt_clause[] di interi con i numeri delle host_vars della clausola
	  //	 lp = intero con la lunghezza della clausola
	  // ..e la aggiunge alla formula
	  // - aggiorno il clause pointer finale	  
	  nclausel++;

	  clause_pointer[NC+1] = clause_pointer[NC]+lp;
	  // - aggiungo la clausola alla formula
	  for(i=0;i<lp;i++)
	    formula[clause_pointer[NC]+i] = learnt_clause[i];
	  // - aggiorno il numero dei letterali e il numero di clausole

	  if (USEGPU%2==1){ // solo se uso gpu
	    HANDLE_ERROR( hipMemcpy( dev_mapped_formula+NL, formula+NL, lp * sizeof(int), hipMemcpyHostToDevice ) );
	    HANDLE_ERROR( hipMemcpy( dev_mapped_cp+NC+1, clause_pointer+NC+1, 1 * sizeof(int), hipMemcpyHostToDevice ) );
	  }

	  NL += lp;
	  NC += 1;

	  if (verbose>1) printf("backjump from level	%d to level %d\n\n\n",level[0],btlevel);
	  level_to_jump=btlevel;	// non chronological backtracking
	  }
	  else {
	    if (verbose>1) printf("backtrack to level %d\n",level[0]);
	    level_to_jump=level[0];	// chronological backtracking
	  }
}


//**************************************************************************
//**************************************************************************
//****** AUXILIARY
//**************************************************************************
//**************************************************************************

__host__ inline void backtrackingf(int level_to_jump){

  // annulla trail stack e riferimenti a variabili
  //  printf("backtrack to %d\n",level_to_jump);

  backtracking++;
  int loop=trail[0]>0;
  while (loop){ // questo mette a posto trail e variabili. level sistemato sulla ricorsione
    int VARP=abs(trail[trail[0]-1]);
    loop=trail[0]>0 &&       
      ((level[VARP]>level_to_jump)   ||  (refs[VARP]>=0)  ||
       (level[VARP]<=level_to_jump && refs[VARP]==-2));
     
    if (level[VARP]>level_to_jump || refs[VARP]>=0 || refs[VARP]==-2){
      //printf("back var %d lev %d\n",VARP,level[VARP]);
      host_vars[VARP] = -1;		// restore "unknown" status
      level[VARP] = -1;
      refs[VARP] = -1;
      trail[0]--;
    }
    else{
      if (level[VARP]<=level_to_jump && refs[VARP]==-1){ // variabile gia' testata -> metto caso opposto
	//printf("switch var %d at lev %d -> %d\n",VARP,level[VARP],-trail[trail[0]-1]);
	refs[VARP] = -2;
	host_vars[VARP] = 1-host_vars[VARP]; // inverse status      
	trail[trail[0]-1]=-trail[trail[0]-1]; // inverto valore su trail
	level[0]=level[VARP];
      }    
    }
  }
} 

__host__ inline void backjumpingf(int level_to_jump){

  // riparte con ultima scelta fatta a level_to_jump
  // annulla trail stack e riferimenti a variabili
  //  printf("backtrack to %d\n",level_to_jump);

  backjump++;
  int loop=trail[0]>0;
  while (loop){ // questo mette a posto trail e variabili. level sistemato sulla ricorsione
    int VARP=abs(trail[trail[0]-1]);
    loop=trail[0]>0 &&       
      ((level[VARP]>level_to_jump)   ||  (refs[VARP]>=0)
       //|| (level[VARP]<=level_to_jump && refs[VARP]==-2)
       );
     
    if (level[VARP]>level_to_jump || refs[VARP]>=0){
      //printf("back var %d lev %d\n",VARP,level[VARP]);
      host_vars[VARP] = -1;		// restore "unknown" status
      level[VARP] = -1;
      refs[VARP] = -1;
      trail[0]--;
    }
    else{ // refs<0 && level=leveltojump
      if (level[VARP]<=level_to_jump && refs[VARP]==-1){ // variabile gia' testata -> metto caso opposto
	//printf("switch var %d at lev %d -> %d\n",VARP,level[VARP],-trail[trail[0]-1]);
	//refs[VARP] = -2;
	//host_vars[VARP] = 1-host_vars[VARP]; // inverse status      
	//trail[trail[0]-1]=-trail[trail[0]-1]; // inverto valore su trail
	level[0]=level[VARP];
      }    
    }
  }
  if (level_to_jump==0)// caso speciale se backjump annulla anche primo livello
    level[0]=0;
} 


//**************************************************************************
//**************************************************************************
//********** MAIN PROCEDURE: twolevel_DPLL *********************************
//********** a part of DPLL (recursive) is handled by the host.  ***********
//********** The final part is made in the device(s) ***********************
//**************************************************************************
//**************************************************************************
//**************************************************************************

__host__  void mask_propagation_cpu(){
  int dbg=0;
    do{ // CPU mask_prop
//INITandy
      mask_prop(&selected_var,&sat_val, &clauind);
//FINEandy
      if (dbg)
       printf("\nDEBUG mask_propagation_cpu:\t mask_prop  sat_val=%d selected_var=%d clauind=%d: ",sat_val,selected_var,clauind);fflush(stdout);
      if (dbg) {
	  printf("\nDEBUG mask_propagation_cpu:\t ");
        for (int i=clause_pointer[clauind];i<clause_pointer[clauind+1];i++)
	  printf(" %d ",formula[i]);
      }
      if (dbg) printf("\n");fflush(stdout);

      if (sat_val==1){
	//	unit++;
	int VAR = abs(selected_var);		// Look for its variable and sign
	if (dbg) printf("\nDEBUG mask_propagation_cpu:\t var prop %d, lev %d\n",selected_var,level[0]); fflush(stdout);
	host_vars[VAR] = selected_var > 0;
	level[VAR] = level[0];
	refs[VAR] = clauind; // devo sapere la clausola
	trail[trail[0]] = selected_var;
	trail[0]++;
      }    
    }
    while (sat_val==1);
      if (dbg) printf("--\n");fflush(stdout);
}

__host__ void mask_propagation_gpu(){
  int dbg=0;
    if (1==0 && dbg){
      printf("at lev %d\n",level[0]);
      for(int i=1;i<trail[0];i++)
	printf("%d@%dr%d,",trail[i],level[abs(trail[i])],refs[abs(trail[i])]);
      printf("\n");
    }
    if (dbg) printf("bl %d, ncl %d NC %d NL %d\n",PARblocks,nclausel,NC,NL);
    do{ // GPU mask_prop
      if (dbg) printf(".\n");
      HANDLE_ERROR( hipMemcpy( dev_parma_vars, host_vars, NV * sizeof(int), hipMemcpyHostToDevice ) );
      parmask_prop<<<PARblocks, THREADS>>>( mask_data, dev_parma_vars, dev_mapped_formula, dev_mapped_cp,NC);
      HANDLE_ERROR( hipMemcpy( h_mask_data, mask_data, (3*PARblocks)*sizeof(int), hipMemcpyDeviceToHost)); // puo' copiare anche 3*blocks+NV per risparmiare chiamate (se servono dati su vars)
      if (dbg) printf("..\n");
      CUDA_count++;
      // colleziona dati da blocchi
      int bestnum=0;
      int bestid=0;                         
      int bestcl=0;                         
      for (int i=0;i<PARblocks;i++){                                
	if (dbg) printf("%d: %d,%d,%d (%d)",i,h_mask_data[3*i],h_mask_data[3*i+1],h_mask_data[3*i+2],bestid);

	if ((h_mask_data[3*i]==-1 && bestnum!=-1) ||
	    (h_mask_data[3*i]==-1 && bestnum==-1 && bestcl>h_mask_data[3*i+2]) ||
	    bestnum==0 ||
	    (bestnum>0 && h_mask_data[3*i]>0 && 
	     (h_mask_data[3*i]<bestnum ||
	      (h_mask_data[3*i]==bestnum && abs(h_mask_data[3*i+1])<abs(bestid)) ||
	      (h_mask_data[3*i]==bestnum && abs(h_mask_data[3*i+1])==abs(bestid) && h_mask_data[3*i+2]<bestcl)
	      ))){
	  bestnum=h_mask_data[3*i];
	  bestid=h_mask_data[3*i+1];
	  bestcl=h_mask_data[3*i+2];
	}    

	/*
	 // TODO: aumenta propagazione in parallelo (una per ogni blocco possibile)
	if (h_mask_data[3*i]==1){
	  int VAR = abs(h_mask_data[3*i+1]);		// Look for its variable and sign
	  if (level[VAR]==-1){ // se non e' gia'stata dedotta da un blocco parallelo (e' sempre -1 grazie a backtracking)
	    if (dbg) printf("var prop %d, lev %d\n",h_mask_data[3*i+1],level[0]);
	    host_vars[VAR] = h_mask_data[3*i+1] > 0;
	    level[VAR] = level[0];
	    refs[VAR] = h_mask_data[3*i+2]; // devo sapere la clausola
	    trail[trail[0]] = h_mask_data[3*i+1];
	    trail[0]++;	
	  }	  
	}
	*/

      }
	if (dbg) printf("\nbest %d %d %d\n",bestnum,bestid,bestcl);
      //printf("\n");
      sat_val  =bestnum;
      selected_var=bestid;
      clauind=bestcl;
      if (dbg) printf("prop %d %d %d: ",sat_val,selected_var,clauind);
      if (dbg)
        for (int i=clause_pointer[clauind];i<clause_pointer[clauind+1];i++)
	  printf("%d ",formula[i]);
      if (dbg) printf("\n");

      if (sat_val==1){
	int VAR = abs(selected_var);		// Look for its variable and sign
	if (dbg) printf("var prop %d, lev %d\n",selected_var,level[0]);
	host_vars[VAR] = selected_var > 0;
	level[VAR] = level[0];
	refs[VAR] = clauind; // devo sapere la clausola
	trail[trail[0]] = selected_var;
	trail[0]++;		  
      }	
    }
    while (sat_val==1);
}


//**********************************************************************************
//**********************************************************************************
//**********************************************************************************
//******************   TWO LEVEL DPLL (no recursive)   *****************************
//**********************************************************************************
//**********************************************************************************
//**********************************************************************************


__host__ int twolevel_DPLL(){
  int selected;
  int pos;
  int FV;
  short good=0;
  int dbg=0;  
  int isbackjumping=0; // a 1 se fatto backj e quindi devo continuare (anche se il bj ha portato il livello a 0 = ho imparato clausola unitaria, e con una UP posso ripartire da livello 1 con l'altro valore)
  do {    

    isbackjumping=0;

    if (dbg) printf("twolevel_DPLL lev %d\n ",level[0]);
    
    // mask_propagation
    if (USEGPU %2 == 0) {
      mask_propagation_cpu();
//printf("TEST 101000\n");
    }
    else 
      mask_propagation_gpu();

  if (dbg){ 
    printf("dopo maskprop lev %d host_vars: ",level[0]);
      for(int i=1;i<trail[0];i++)
	printf("%d@%dr%d,",trail[i],level[abs(trail[i])],refs[abs(trail[i])]);
      printf("\n");
      /*    for (int i=0;i<NV;i++)
      printf("%d>%d(%d,%d) ",i,host_vars[i],level[i],refs[i]);
    printf("\n");
      */
    printf("sat_val %d, selvar %d\n",sat_val,selected_var);     
  }   

//printf("TEST 102000\n");
  // new case
  if (level[0]<=0 && sat_val<0){ // failed, UP was enough, no need to learn, return fail!
    return 0;
  }
  
  //************* UNSATISFIABLE ASSIGNMENT
  if (sat_val < 0){ // At least one clause is false
//printf("TEST 103000\n");
    good = 0;
    refs[0] = clauind;	// conflict clause
    if (learning) {// CLAUSE LEARNING
      clause_learning(); 
      if (NC%1000 ==0) printf("NC: %d\n",NC);
      if (dbg) printf("cl: level to backjump to %d\n",level_to_jump);      
      if (dbg) printf("backjump %d\n",level_to_jump);
      backjumpingf(level_to_jump);
      isbackjumping=1;
//printf("TEST 104000\n");
    }
    else{
      if (dbg) printf("backtrack %d\n",level[0]);
      backtrackingf(level[0]); // se non imparo, apro fratello (tengo livello corrente)
    }

    // END CLAUSE LEARNING
  }  
  else if (sat_val == 0){  //************* FOUND A SOLUTION
      good = 1;  
  }      
  else    
  { // There is a non-ground literal - sat_val > 0
//printf("TEST 105000\n");
    pos = (selected_var > 0);  
    selected = abs(selected_var);  // Look for its variable and sign 
    FV = NV-trail[0]; // free vars
    if (USEGPU >= 2 && FV <= MaxV)   { //*** GPU CALL: 
//      printf("CUDA caller with %d learnt clauses\n",nclausel);
      good=CUDA_caller();
      if (!good)
	backtrackingf(level[0]); //backtrack
//printf("TEST 106000\n");
    }   
    else { ///////// vado con CPU
//printf("TEST 107020\n");
      level[0]++; // apro nuovo ramo
      if (dbg)
	printf("lev %d, 2 scelte: var %d val %d\n",level[0],selected,pos);
//printf("TEST 107030\n");
      level[selected] = level[0];
//printf("TEST 107040\n");
      host_vars[selected] = pos;
//printf("TEST 107050 trail[0]=%d  selected_var=%d\n",trail[0],selected_var);
      trail[trail[0]++] = selected_var;
//printf("TEST 108000\n");
    } // CPU
  } //sat_val >2


  }
  while(isbackjumping || (level[0]>0 && !good));
  return good;
}



//************************************************************************
//************************************************************************
//************************************************************************
// NEW: Version with watched literals (option 4)
//************************************************************************
//************************************************************************
//************************************************************************

int  unit_propagation(
   int* U, int* UNIT, int* V2C, 
   int* W, int* NEXT, int** C2V,
   int* top, int** varstack,
   int* nextselected){
   
   int n_p,temp,c,v, lit,previous;
  
   // printf("#### ENTERED IN UNIT PROPAGATION\n");

   *nextselected = 0;
   while(*U >= 0){
      // *** v is the current (deterministic) variable
      v = abs(UNIT[*U]);
      if (host_vars[v] < 0){
          host_vars[v] = (UNIT[*U] > 0);
          (*top)++;
          varstack[*top][0] = UNIT[*U]; //*** signed literal
          varstack[*top][1] = 2 ; //*** 2 -> not backtrackable 
	  //printf("CASE 4 - UP var[%d]\n",abs(UNIT[*U]));
      }    
      (*U)--; //*** v removed from the unit list.

      n_p = V2C[v];     
      while (n_p > -1){ //*** Visit the v list (starting from V2C[v])       
         c = W[n_p];
         lit = clause_pointer[c];
         //*** It is not sufficient to check only the "v" based  literal!
         //*** Could be sped up with a further (backtrackable) array
         temp=1;
         while(temp && (lit < clause_pointer[c+1]))
             if ( (host_vars[abs(formula[lit])] >= 0) && ((formula[lit] > 0) == host_vars[abs(formula[lit])]))
                 temp=0;
             else lit++;
         //*** check if v satisfies the clause
         if (!temp) {
            // printf(">>> Clause %d satisfied by current assignment\n",c);
            previous = n_p;
            n_p  = NEXT[previous];
        }   
        else { //*** clause NOT satisfied by the v assignment 
               //*** seek for one unassigned literal (restart from beginning of the clause)
             // printf(">>> Clause %d **NOT** satisfied by current assignment\n",c);
             lit = clause_pointer[c];
             while((lit < clause_pointer[c+1]) && (host_vars[abs(formula[lit])] >= 0))  lit++;
             // *** Clause c is false: backtracking needed.
             if (lit == clause_pointer[c+1]){ 
                 // printf(">>> Exit due to failure\n");
                 *U = -1; // Empty the Unit list
                 return 0;
              }      
             //*** We know that formula[lit] is satisfiable. We know abs(formula[lit]) is not v   
             // *** if the first non ground literal found is already watching c, seek another
             if (abs(formula[lit]) == C2V[c][0] || abs(formula[lit]) == C2V[c][1]){
                 temp=formula[lit];
                 lit++;
                 while((lit < clause_pointer[c+1]) && (host_vars[abs(formula[lit])] >= 0)) lit++;
                // *** if there is not another:  UNIT CASE: add it to unit and update pointers
                if (lit == clause_pointer[c+1]){  
                    // printf("Aggiungo %d a UNIT\n", temp);
                    (*U)++;
                    UNIT[*U] = temp;
                    previous = n_p;
                    n_p = NEXT[previous];
                 }                     
              } // if the first literal is watching   
              
              if (lit < clause_pointer[c+1]){ //*** Update in the non unit case
                  // **** SELECTION STRATEGY: the LAST found non ground watched literal different 
                     *nextselected = formula[lit]; // from v in the LAST non satisfied clause visited 
                     //printf("SELECTED literal %d\n",formula[lit]);
                     //*** LISTS REARRANGEMENTS
                    if (C2V[c][0]==v) C2V[c][0] = abs(formula[lit]);
                       else  C2V[c][1] = abs(formula[lit]);
                    temp = V2C[abs(formula[lit])];
                    if (n_p == V2C[v]){  //*** We are at the beginning of the list 
                       V2C[v] = NEXT[n_p];
                       NEXT[n_p] = temp;
                       V2C[abs(formula[lit])] = n_p;
                       n_p = V2C[v];
                    }     
                    else {
                       NEXT[previous] = NEXT[n_p];                             
                       V2C[abs(formula[lit])] = n_p;
                       NEXT[n_p] = temp;
                       n_p = NEXT[previous];
                    }
              } // Update in the non unit casa      
          } // end if clause not satisfied by "v" assignment
      } // visit the v list
    }  // while *U      

    return 1;
}        
        
//*** AUX. Data Structure Initialization

void insert(int c, int v, int p_W, int zerone, int* V2C, 
            int* W, int* NEXT, int** C2V){
     int ptemp = V2C[v];
     V2C[v]         = p_W;     W[p_W]         = c;
     NEXT[p_W]      = ptemp;   C2V[c][zerone] = v;
}     

//*** AUX. Choice of a variable
//*** unit_propagation in some cases do not select properly a literal
//*** (e.g. in the case of different connected components)

int first_seed(int* host_vars, int* V2C){
    int i=1,temp=1;
    
     while(temp)
          if( (i < NV) && (host_vars[i] < 0) && (V2C[i] >= 0) )
                temp = 0;
          else i++;       
     if (i < NV) 
         return i; //*** Found a non ground literals pointing
     else {
         i=1;
         temp=1;    
         while(temp)
              if( (i < NV) && (host_vars[i] < 0) )
                temp = 0;
               else i++;
         if (i < NV)  
               return i; //*** Found a non ground literal not pointing
         else return 0;
    } 
} 


//************************************************************************
// DPLL with watched literals
//************************************************************************

__host__ int watchedDPLL(){
  int i, U,sat,top, proceed, SEL=0, temp;
  int* V2C, *W, *NEXT, *UNIT, ** C2V, **varstack;
  
//*** DYNAMIC DATA STRUCTURES BORING DEFINITION
//*** From each variable V to a list of clauses guarded by V
  V2C  =  (int*) malloc(2 * NV * sizeof(int));
//*** Lists are stored here:  
  W    =  (int*) malloc(2 * NC * sizeof(int));
  NEXT =  (int*) malloc(2 * NC * sizeof(int));
//*** Temporary list of literals ready for unit-propagation:  
  UNIT = (int*) malloc(NV * sizeof(int));   
//*** Backward pointers: from a clause to the pair of current watched literals (matrix NC x 2)  
  C2V =  (int**) malloc(NC * sizeof(int*)); 
//*** The data structure for recursion  NC x 2
  varstack = (int**) malloc(NC * sizeof(int*));
//*** Rows allocation.
  for(i=0;i<NC;i++){
     varstack[i] =  (int*) malloc(2 * sizeof(int));
     C2V[i]  =  (int*) malloc(2 * sizeof(int));
  }
  
//*** RESET OF THE WATCHED STRUCTURE:

  for(i=0;i<2*NC;i++){ W[i]   = -1;  NEXT[i] = -1; }
  for(i=0;i<NC;i++){ C2V[i][0] = 0;  C2V[i][1] = 0; }
  for(i=0;i<NV;i++)  { V2C[i] = -1;  UNIT[i] = 0;  }
  U=-1;  top=-1;  sat=1; SEL=0;
  
//*** INITIALIZATION OF WATCHED MATRIX

  for(i=0;i<NC;i++){
      //*** FIRST LITERAL WATCHED
      insert(i,abs(formula[clause_pointer[i]]),2*i,0,V2C,W,NEXT,C2V);
      //*** CHECK THAT THERE ARE AT LEAST TWO LITERALS IN THE CLAUSE
      if (clause_pointer[i+1] > clause_pointer[i] + 1){
      //*** SECOND LITERAL WATCHED (IF ANY)
         insert(i,abs(formula[clause_pointer[i]+1]),2*i+1, 1, V2C,W, NEXT,C2V);
      } else { //*** READY TO APPLY UNIT PROPAGATION FOR THIS LITERAL (no repetition control!!! - this might cause error)
         U++;
         UNIT[U] = formula[clause_pointer[i]]; // Positive or negative
      }
   }   
   
  
  /*
   printf("**** DATA STRUCTURE *****************************************\n");  
   for(i=0;i<NC;i++)
     printf("Watched clausola %d = %d, %d\n", i, C2V[i][0], C2V[i][1]);
   for(i=0;i<2*NC;i++)
     printf("Linked structure i=%d, (%d,%d) \n", i, W[i], NEXT[i]);
   for(i=1;i<NV;i++)
     printf("Var pointer V2C[%d] = %d\n", i, V2C[i] );
   printf("**********************************************************\n");  
  */
      sat = unit_propagation(&U,UNIT,V2C,W,NEXT,C2V,&top,varstack,&SEL);
      
/*
   printf("EXITED U.P. with U=%d, SEL=%d, top=%d, sat=%d\n", U, SEL, top, sat);
   for(i=1; i <NV; i++)printf("v[%d]=%d",i,host_vars[i]);
   printf("\n");
*/

      if (sat == 0) return 0;    // *** EXIT WITH FAILURE      
      
      if (top == NV-2) return 1; // *** EXIT WITH SUCCESS (trivial instance)

      // Heuristic first seed, if not set by unit_propagation
      
      if (SEL == 0) SEL = first_seed(host_vars,V2C); 

      // *** Initialize the stack with a backtrackable var      
      top++;
      varstack[top][0]= SEL;   // E.g. v satisfied by 1 -> -1
      varstack[top][1]= (SEL>0)-2; // -v satisfied by 0 -> -2     
                       
//****************************************************************************
//*** MAIN LOOP
//****************************************************************************
      
      int dbg=0;

      while((top >=0) && (top < NV-1)){
	proceed = 0;
	while((top >= 0) && !proceed){  
	  if (varstack[top][1] > 1){ // *** no longer  backtrackable
	    if (dbg) printf("CASE 1 - pop of var[%d]-\n",abs(varstack[top][0]));
	    host_vars[abs(varstack[top][0])] = -1;
	    varstack[top][1] = -1;
	    top--;// Restore unknown status              
	  } else if (varstack[top][1] < 0) {// Never tried value. Try and update
	    if (dbg)  printf("CASE 2 - 1st attempt for var[%d]=%d\n",abs(varstack[top][0]),2+varstack[top][1]);
	    proceed = 1;
	    host_vars[abs(varstack[top][0])] = 2+varstack[top][1]; 
	    varstack[top][1] = host_vars[abs(varstack[top][0])];
	  } else if (varstack[top][1] < 2) {// Last chance
	    if (dbg)  printf("CASE 3 - 2nd attempt  for var[%d]=%d\n",abs(varstack[top][0]),1-varstack[top][1]);
	    proceed = 1;
	    host_vars[abs(varstack[top][0])] = 1-varstack[top][1];
	    varstack[top][1] = 2;
	  } 
        }// *** while (! proceed)
             
	if (top >= 0){
	  U++;
	  UNIT[U]= varstack[top][0];
	  //*** CALL TO UNIT PROPAGATION  
	  sat = unit_propagation(&U,UNIT,V2C,W,NEXT,C2V,&top,varstack,&SEL);
	  //printf("sat %d, top %d, NV %d\n",sat,top,NV);

	  // ALE: check sat, cosi' esce subito (novita' per non creare problemi all'aggancio con GPU)
	  if (sat && (top >=NV-2))
	    return 1;

	  if ( (NV-1-top-1<MaxV) && USEGPU==5){ // uso cudadpll_w
	    if (sat && (top < NV -2) && NV-1-top-1>0){ // almeno una FV (anche se io eviterei proprio di chiamare...)
	      //printf("CUDA CALLER: free vars %d, %d %d\n",NV-1-top-1,NV,top);
	      sat=CUDA_caller_watched();
	      if (sat)
		return sat; // altrimenti prosegue come se sat falso (backtracking)
	    }
	    else{ // qui entra se non sat (e backtrack), oppure sat, ma con 0 FV e quindi non chiama cudacaller
	      //	      printf("sat %d\n",sat);
	    }
	  }
	  else 
	      {
	      if (sat && (top < NV -1) && !SEL){
		//*** SEL becomes the next entry for U.P.
		i=1; temp=1;
		while(temp && (i < NV))
		  if( (host_vars[i] < 0) )  
		    temp = 0;
		  else i++;
		SEL=i;
	      } 
              
	      if (sat && SEL)  {
		top++;
		varstack[top][0]=SEL;
		varstack[top][1]=(SEL >0) -2;     
	      }
	      }
	} // if (top>=0)      
      } // *** main loop     
      printf("sat %d\n",sat);
      return sat;
}  


//************************************************************************
//************************************************************************
//*******************       main        **********************************
//************************************************************************
//************************************************************************

__host__ int  main(int argc, char** argv) {
    int retv = 0;
    hipDeviceProp_t prop;
    int whichDevice;
    
//INITandy
    NUMPARAMETERS = 9;
    if (argc!=NUMPARAMETERS){
      printf("usage: %s mode N_vars_blocks(2LOG_BLKS) N_vars_thread(LOG_THRDS) Maxv learning(0/1) varSelectStrategyCPU(0-4) varSortGPU(0/1) filename\n",argv[0]);
      return -1;
    }    
//FINEandy
    
    // *** PARAMETER SETTINGS
    USEGPU      = atoi(argv[1]);
    LOG_BLKS    = atoi(argv[2]);
    LOG_THRDS   = atoi(argv[3]);
    CUDABLOCKS  = (1<<LOG_BLKS);   //*** pow(2,LOG_BLKS); 
    CUDATHREADS = (1<<LOG_THRDS);  //*** pow(2,LOG_THRDS);  
    MaxV        = atoi(argv[4]);  
    Delta       = max(1,MaxV - LOG_BLKS + 1); // *** In case MaxV < LOG_BLKS use default "1"  
    learning    = atoi(argv[5]);
//INITandy
    strategia_varSelHost    = atoi(argv[6]);
    strategia_maps_varSort    = atoi(argv[7]); //usa strategia_varSelHost anche per riordino di maps_vars (1 yes, <>1 no)
//FINEandy

    if (USEGPU>0){
      // CUDA timings initialization
      hipEventCreate( &start );
      hipEventCreate( &stop );
      // Checking of CUDA HW capabilities
      HANDLE_ERROR( hipGetDevice( &whichDevice ) );
      HANDLE_ERROR( hipGetDeviceProperties( &prop, whichDevice ) );
      if (prop.canMapHostMemory != 1) {
	            printf( "Device cannot map memory.\n" );
	  return 0;
      }
    }

    //*** Some "warning" and exit
    if(USEGPU>0 && CUDATHREADS > prop.maxThreadsPerBlock){
      printf("Cant handle so many vars per block (max %d < %d threads per block)\n",CUDATHREADS,prop.maxThreadsPerBlock);
      return -1;
    }
    printf("using LOG_BLKS=%d, LOG_THRDS=%d MaxV=%d \n",LOG_BLKS,LOG_THRDS,MaxV);
    if (Delta > DV_SIZE){
        printf("Delta too large, recompile kernel with %d in the size of arrays\n",Delta);
        return -1;
    }    
    
    //**** CORE COMPUTATION
     allocate_first();
    // *** Load a SAT formula in DIMACS format

     load_formula(argv[NUMPARAMETERS-1],&NV,&NC,&NL);

    if (USEGPU %2 == 0) {
    	printf("INFO: Variable selection strategy (host): ");
    	print_option(strategia_varSelHost); printf("\n");
    }

    //*** Print and use some HW and instance info     
     if (USEGPU>0){
       print_info(prop);
       PARblocks = IMIN( ( NC + THREADS - 1) / THREADS, 2*prop.multiProcessorCount );
     } 
    
     allocate_second();
    //*** Start running time (formula loading is not counted)    
     if (USEGPU %2 == 1 && USEGPU<4)
         printf("blocks %d, threads %d\n",PARblocks, THREADS);
     srand ( time(NULL) );
     printf("start\n");
     
     if (USEGPU>0){
       hipEventRecord(start,0);
       hipEventSynchronize(start);    
     }else{
#if (defined(_WIN32) || defined(__WIN32__) || defined(WIN32))
       global_time=clock();
#else
       gettimeofday(&time_start,NULL);
#endif
     }

//*********************************************************************
//******* Call the main procedure (store the result in vars[0]): 
//*********************************************************************
      
     if (USEGPU < 4){      
       retv = twolevel_DPLL();
       host_vars[0] = retv;  
     }
     else // for mode 4 and 5
       host_vars[0] = watchedDPLL();
    
    //*** CUDA timings:
    if (USEGPU>0){
      hipEventRecord( stop, 0 );    
      hipEventSynchronize( stop );    
      hipEventElapsedTime( &deltatime, start, stop );
    }
    else{
#if (defined(_WIN32) || defined(__WIN32__) || defined(WIN32))
     clock_t final=clock()-global_time;
     deltatime=((float)final/CLK_TCK)*1000;
#else
     gettimeofday(&time_stop,NULL);
     deltatime=(0.0+time_stop.tv_sec+time_stop.tv_usec/1000000.0) - (0.0+time_start.tv_sec+time_start.tv_usec/1000000.0);
     deltatime*=1000;
#endif
    }

//printf("TEST 5000\n");
    /* DISPLAY RESULT AND RUNNING TIME: */
    print_result( host_vars, NV );    
    print_time( deltatime );
             
    deallocate();
    return 0;
}            

//********* END *******************************************************
